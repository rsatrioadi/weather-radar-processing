#include "hip/hip_runtime.h"
#include <stdio.h>
#include <complex.h>

#include <hip/hip_complex.h>

#include "cumatrixtools.h"
#include "cuinverse_kernel.cu"

void cuPrintMatrix( hipFloatComplex *C ,int N, int M ) {
    int i,j;
    for(i=0;i<N;i++) {
        for(j=0;j<M;j++) 
            printf(" (%f,%f)\t ", hipCrealf(C[i*N + j]) , hipCimagf(C[i*N + j]) );
        printf(" \n ");
    }
}

///////////////////////////////////////////////////////////////////////////////
const int N=3; //Matrix dimension

const int NBranches = 64;
///////////////////////////////////////////////////////////////////////////////
// Main program
///////////////////////////////////////////////////////////////////////////////
int main(int argc, char **argv){

    int i;
    float complex A[9] = // Base matrix
    {

        0.f+I/sqrtf(2.f), 0.0f+ I/sqrt(2.0f), 0.+ 0.*I,

        0. -I/2. , 0.+I/2 , 0.+ I/sqrt(2.),
        -0.5+0.*I , 0.5+0.*I , -1./sqrt(2.)+.0*I

    };

    hipFloatComplex *h_A, *h_invA;
    hipFloatComplex *d_A, *d_invA, *d_WorkSpace;


    printf("...allocating CPU memory.\n");
    h_A = (hipFloatComplex *) malloc( N*N*sizeof(hipFloatComplex ));
    h_invA = (hipFloatComplex *) malloc( NBranches*N*N*sizeof(hipFloatComplex ));

    printf("...allocating GPU memory.\n");

    hipMalloc((void **)&d_A, NBranches*N*N*sizeof(hipFloatComplex ));

    hipMalloc((void **)&d_invA, NBranches*N*N*sizeof(hipFloatComplex ));

    hipMalloc((void **)&d_WorkSpace, NBranches*cgeMatrixInverse_WorkSpace()*N*N*sizeof(hipFloatComplex ));


    printf("...Copying memory.\n ");
    for(i=0;i<N*N;i++ ) 
        h_A[i] = make_hipFloatComplex( crealf(A[i]) , cimagf(A[i]) );

    hipMemcpy(d_A, h_A, N*N*sizeof(hipFloatComplex) , hipMemcpyHostToDevice);


    printf("...The base matrix is:\n");
    cuPrintMatrix( h_A , N, N );


    printf("\n...Calling the kernel.\n");

    hipDeviceSynchronize();
    cgeMatrixInverse_kernel<<<2,32>>>(d_invA, d_A , N ,d_WorkSpace); // Divinding the 64 branches in 2 blocks of 32 threads

    hipDeviceSynchronize();

    hipMemcpy(h_invA, d_invA, NBranches*N*N*sizeof(float), hipMemcpyDeviceToHost);


    printf("\n The inverse of the first branch is \n");
    cuPrintMatrix( h_invA , N, N );


    printf("\n The inverse of the second branch is \n");
    cuPrintMatrix( h_invA + N*N , N, N );


    printf("\n and so on ..\n");

    free(h_A);
    free(h_invA);


    hipFree(d_A);
    hipFree(d_invA);

    hipDeviceReset();
    printf("\n-------------------------------------------------------\n");
}