#include "hip/hip_runtime.h"
#include <math.h>
#include <iostream>
#include "hip/hip_runtime.h"
#include <stdlib.h>

using namespace std;

__global__ void __apply_hamming(hipDoubleComplex *a, double *b, int m, int n) {

    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;

    double real, imag;
    real = a[i].x;
    imag = a[i].y;
    a[i] = make_hipDoubleComplex(b[i%(m*n)]*real, b[i%(m*n)]*imag);
}

/*hipDoubleComplex *pcmul_gpu(hipDoubleComplex *a, double *b, int m, int n) {

    // host
    hipDoubleComplex *h_res;
    // device
    hipDoubleComplex *d_a;
    double *d_b;

    // struct timeval tb, te;
    // unsigned long long bb, e;

    h_res = new hipDoubleComplex[2*m*n];

    hipMalloc(&d_a, 2*m*n*sizeof(hipDoubleComplex));
    hipMalloc(&d_b, m*n*sizeof(double));

    // gettimeofday(&tb, NULL);

    hipMemcpy(d_a, a, 2*m*n*sizeof(hipDoubleComplex), hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, m*n*sizeof(double), hipMemcpyHostToDevice);

    // gettimeofday(&te, NULL);
    // bb = (unsigned long long)(tb.tv_sec) * 1000000 + (unsigned long long)(tb.tv_usec) / 1;
    // e = (unsigned long long)(te.tv_sec) * 1000000 + (unsigned long long)(te.tv_usec) / 1;

    // cout << "copy to device " << e-bb << endl;

    // gettimeofday(&tb, NULL);
    pcmul_kernel<<<2*m,n>>>(d_a, d_b, m, n);
    // gettimeofday(&te, NULL);
    // bb = (unsigned long long)(tb.tv_sec) * 1000000 + (unsigned long long)(tb.tv_usec) / 1;
    // e = (unsigned long long)(te.tv_sec) * 1000000 + (unsigned long long)(te.tv_usec) / 1;

    // cout << "kernel compute " << e-bb << endl;

    // gettimeofday(&tb, NULL);
    hipMemcpy(h_res, d_a, m*n*sizeof(hipDoubleComplex), hipMemcpyDeviceToHost);
    // gettimeofday(&te, NULL);
    // bb = (unsigned long long)(tb.tv_sec) * 1000000 + (unsigned long long)(tb.tv_usec) / 1;
    // e = (unsigned long long)(te.tv_sec) * 1000000 + (unsigned long long)(te.tv_usec) / 1;

    // cout << "copy to host " << e-bb << endl;

    hipFree(d_a);
    hipFree(d_b);

    return h_res;
}*/
