#include <iostream>
#include <stdlib.h>
#include <time.h>
#include <hip/hip_runtime.h>
#include <math.h>
#include <complex.h>

using namespace std;

complex<float> *pcmul(complex<float> *a, float *b, int m, int n) {
    complex<float> *res = new complex<float>[m*n];

    for (int i=0; i<m; i++) {
        for (int j=0; j<n; j++) {
            res[i*n+j] = b[i*n+j] * a[i*n+j];
        }
    }

    return res;
}

complex<float> *pcmul_gpu(complex<float> *a, float *b, int m, int n) {
    
    complex<float> *res = new complex<float>[m*n];

    hipFloatComplex *d_a, *d_res, *h_a, *h_res;
    float *d_b;

    h_a = new hipFloatComplex[m*n];
    h_res = new hipFloatComplex[m*n];
    hipMalloc(&d_a, m*n*sizeof(hipFloatComplex));
    hipMalloc(&d_b, m*n*sizeof(float));
    hipMalloc(&d_res, m*n*sizeof(hipFloatComplex));

    for (int i=0; i<m; i++) {
        for (int j=0; j<n; j++) {
            h_a[i*n+j] = make_hipFloatComplex(a[i*n+j].real(), a[i*n+j].imag());
        }
    }

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_res);
    delete[] h_res;

    return res;
}

int main(int argc, char **argv) {

    int m, n;
    complex<float> *iq, *mul;
    float *h;

    m = 8;
    n = 4;

    iq = new complex<float>[m*n];
    h = new float[m*n];

    for (int i=0; i<m; i++) {
        for (int j=0; j<n; j++) {
            iq[i*n+j] = {(float)i,(float)j};
            h[i*n+j] = i;
            cout << iq[i*n+j] << " ";
        }
        cout << endl;
    }

    for (int i=0; i<m; i++) {
        for (int j=0; j<n; j++) {
            cout << h[i*n+j] << " ";
        }
        cout << endl;
    }

    mul = pcmul(iq, h, m, n);

    for (int i=0; i<m; i++) {
        for (int j=0; j<n; j++) {
            cout << mul[i*n+j] << " ";
        }
        cout << endl;
    }

    delete[] iq;
    delete[] h;
    delete[] mul;

    return 0;
}
