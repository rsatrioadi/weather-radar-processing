#include "hip/hip_runtime.h"
__global__ void cgeMatrixInverse_kernel(hipFloatComplex *invA , hipFloatComplex *A , int N , hipFloatComplex *Work) {

    int i;
    int idx = blockDim.x * blockIdx.x + threadIdx.x;

    hipFloatComplex * ThreadWorkSpace;

    ThreadWorkSpace = Work + idx*cgeMatrixInverse_WorkSpace()*N*N;

    for(i=0; i<N*N; i++) 
        A[ i + idx*N*N ] =  A[i];

    A[ idx*N*N ] = make_hipFloatComplex( (float) idx , 1./sqrtf(2));

    cgeMatrixInverse(invA + idx*N*N , A + idx*N*N , N , ThreadWorkSpace);
}