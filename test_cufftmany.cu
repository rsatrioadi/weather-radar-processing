#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>
#include <stdio.h>
#include <math.h>
#include <iostream>

#define DATASIZE 4
#define BATCH 8

/********************/
/* CUDA ERROR CHECK */
/********************/
#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true) {
   if (code != hipSuccess) {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

/********/
/* MAIN */
/********/
int main () {
    // --- Host side input data allocation and initialization
    hipfftReal *hostInputData = (hipfftReal*)malloc(DATASIZE*BATCH*sizeof(hipfftReal));
    for (int i=0; i<BATCH; i++)
        for (int j=0; j<DATASIZE; j++) 
            std::cin >> hostInputData[i*DATASIZE + j];

    // --- Device side input data allocation and initialization
    hipfftReal *deviceInputData; gpuErrchk(hipMalloc((void**)&deviceInputData, DATASIZE * BATCH * sizeof(hipfftReal)));
    hipMemcpy(deviceInputData, hostInputData, DATASIZE * BATCH * sizeof(hipfftReal), hipMemcpyHostToDevice);

    // --- Host side output data allocation
    hipfftComplex *hostOutputData = (hipfftComplex*)malloc((DATASIZE / 2 + 1) * BATCH * sizeof(hipfftComplex));

    // --- Device side output data allocation
    hipfftComplex *deviceOutputData; gpuErrchk(hipMalloc((void**)&deviceOutputData, (DATASIZE / 2 + 1) * BATCH * sizeof(hipfftComplex)));

    // --- Batched 1D FFTs
    hipfftHandle handle;
    int rank = 1;                           // --- 1D FFTs
    int n[] = { DATASIZE };                 // --- Size of the Fourier transform
    int istride = 1, ostride = 1;           // --- Distance between two successive input/output elements
    int idist = DATASIZE, odist = (DATASIZE / 2 + 1); // --- Distance between batches
    int inembed[] = { 0 };                  // --- Input size with pitch (ignored for 1D transforms)
    int onembed[] = { 0 };                  // --- Output size with pitch (ignored for 1D transforms)
    int batch = BATCH;                      // --- Number of batched executions
    hipfftPlanMany(&handle, rank, n, 
                  inembed, istride, idist,
                  onembed, ostride, odist, HIPFFT_R2C, batch);

    //cufftPlan1d(&handle, DATASIZE, CUFFT_R2C, BATCH);
    hipfftExecR2C(handle,  deviceInputData, deviceOutputData);

    // --- Device->Host copy of the results
    gpuErrchk(hipMemcpy(hostOutputData, deviceOutputData, (DATASIZE / 2 + 1) * BATCH * sizeof(hipfftComplex), hipMemcpyDeviceToHost));

    for (int i=0; i<BATCH; i++)
        for (int j=0; j<(DATASIZE / 2 + 1); j++)
            printf("%i %i %f %f\n", i, j, hostOutputData[i*(DATASIZE / 2 + 1) + j].x, hostOutputData[i*(DATASIZE / 2 + 1) + j].y);

    hipfftDestroy(handle);
    gpuErrchk(hipFree(deviceOutputData));
    gpuErrchk(hipFree(deviceInputData));
}
