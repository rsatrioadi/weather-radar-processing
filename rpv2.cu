#include "hip/hip_runtime.h"
#include <iostream>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hip/hip_complex.h>
#include <fftw3.h>
#include <hipfft/hipfft.h>
#include <sys/time.h>
#include <assert.h>
#include "floats.h"
#include "dimension.h"

#include <msgpack.hpp>
#include "zhelpers.hpp"

#define NUM_BYTES_PER_SAMPLE (3*2*2)

using namespace std;

struct sectormsg {
  int id;
  vector<int> 
      i_hh, q_hh,
      i_vv, q_vv,
      i_vh, q_vh;
  MSGPACK_DEFINE_MAP( id, i_hh, q_hh, i_vv, q_vv, i_vh, q_vh );
};

const int
    n_sectors = 143,
    n_sweeps = 1024,
    n_samples = 512,
    n_elevations = 9;
static const int k_range_resolution = 30;
static constexpr float k_calibration = 1941.05;
static const int ma_count = 7;
int
    current_sector = 0,
    current_sweep = 0,
    current_sample = 0,
    current_elevation = 0,
    current_stream = 0;

int
    hh_index_start,
    vv_index_start,
    vh_index_start,
    input_stream_index_offset;

// host
hipFloatComplex *p_iq;
float *result;
float *hamming_coef;
hipFloatComplex *fft_ma;

// device
float *d_hamming;
hipFloatComplex *d_iq;
hipFloatComplex *d_tmp;
float *d_result;

// cufft
hipfftHandle
    *fft_range_handle,
    *fft_doppler_handle,
    *fft_pdop_handle;

hipStream_t *streams;

zmq::context_t context( 1 );
zmq::socket_t subscriber( context, ZMQ_SUB );
zmq::socket_t publisher( context, ZMQ_PUB );


__constant__ hipFloatComplex d_ma[512];

__global__ void __apply_hamming(hipFloatComplex *a, float *b, int offset) {
  const unsigned int idx = threadIdx.x + blockIdx.x*blockDim.x;
  a[offset + idx] = make_hipFloatComplex(
      b[idx]*hipCrealf( a[offset + idx] ),
      b[idx]*hipCimagf( a[offset + idx] ));
}

__global__ void __sum_v4(hipFloatComplex *in, hipFloatComplex *out, int offset) {
  const unsigned int i = 2*blockIdx.x, j = threadIdx.x, n = blockDim.x;
  extern __shared__ hipFloatComplex sdata[];

#pragma unroll
  for (unsigned int d = 0; d < 2; d++) {
    sdata[j + n*d] = make_hipFloatComplex(
        in[offset + j + i*n + n*d].x,
        in[offset + j + i*n + n*d].y );
  }
  __syncthreads();

  for (unsigned int s = blockDim.x/2; s > 0; s >>= 1) {
    if (j < s) {
#pragma unroll
      for (unsigned int d = 0; d < 2; d++) {
        sdata[j + n*d] = hipCaddf( sdata[j + n*d], sdata[j + n*d + s] );
      }
    }
    __syncthreads();
  }

  if (j == 0) {
#pragma unroll
    for (unsigned int d = 0; d < 2; d++) {
      out[i*n + n*d] = sdata[j + n*d];
    }
  }
}

__global__ void __avgconj(hipFloatComplex *inout, hipFloatComplex *sum, int offset) {
  const unsigned int i = blockIdx.x, j = threadIdx.x, n = blockDim.x;

  float avgx = sum[offset + i*n].x/n;
  float avgy = sum[offset + i*n].y/n;
  inout[offset + j + i*n] = make_hipFloatComplex( inout[offset + j + i*n].x - avgx,
                                                 (inout[offset + j + i*n].y - avgy)*-1 );
}

__global__ void __conjugate(hipFloatComplex *a, int offset) {
  const unsigned int idx = blockIdx.x*blockDim.x + threadIdx.x;
  a[offset + idx].y *= -1;
}

__global__ void __shift(hipFloatComplex *inout, int n, int offset) {
  const unsigned int i = blockIdx.x, j = threadIdx.x;

  hipFloatComplex temp = inout[offset + j + i*n];
  inout[offset + j + i*n] = inout[offset + (j + n/2) + i*n];
  inout[offset + (j + n/2) + i*n] = temp;
}

__global__ void __clip_v2(hipFloatComplex *inout, int n, int offset) {
  const unsigned int i = threadIdx.x, j = n - blockIdx.x - 1;
  inout[offset + j + i*n] = make_hipFloatComplex( 0, 0 );
}

__global__ void __abssqr(hipFloatComplex *inout, int offset) {
  const unsigned int idx = blockIdx.x*blockDim.x + threadIdx.x;

  float real, imag;
  real = hipCrealf( inout[offset + idx] );
  imag = hipCimagf( inout[offset + idx] );
  inout[offset + idx] = make_hipFloatComplex( real*real + imag*imag, 0 );
}

__global__ void __apply_ma(hipFloatComplex *inout, int offset) {
  const unsigned int i = blockIdx.x, j = threadIdx.x, n = blockDim.x;

  inout[offset + j + i*n] = hipCmulf( inout[offset + j + i*n], d_ma[j] );
}

__global__ void __scale_real(hipFloatComplex *inout, int offset) {
  const unsigned int i = blockIdx.x, j = threadIdx.x, n = blockDim.x;

  inout[offset + j + i*n] = make_hipFloatComplex( inout[offset + j + i*n].x/n, 0 );
}

__global__ void __sum_inplace_v4(hipFloatComplex *in, int offset) {
  const unsigned int i = 2*blockIdx.x, j = threadIdx.x, n = blockDim.x;
  extern __shared__ hipFloatComplex sdata[];

#pragma unroll
  for (unsigned int d = 0; d < 2; d++) {
    sdata[j + n*d] = make_hipFloatComplex( in[offset + j + i*n + n*d].x, in[offset + j + i*n + n*d].y );
  }
  __syncthreads();

  for (unsigned int s = blockDim.x/2; s > 0; s >>= 1) {
    if (j < s) {
#pragma unroll
      for (unsigned int d = 0; d < 2; d++) {
        sdata[j + n*d] = hipCaddf( sdata[j + n*d], sdata[j + n*d + s] );
      }
    }
    __syncthreads();
  }

  if (j == 0) {
#pragma unroll
    for (unsigned int d = 0; d < 2; d++) {
      in[offset + (i*n + n*d)] = sdata[j + n*d];
    }
  }
}

__global__ void __calcresult_v2(
    hipFloatComplex *iq,
    float *out,
    int n,
    int offset_hh, int offset_vv, int offset_vh,
    int result_offset) {

  const unsigned int i = threadIdx.x;

  float z = pow( i*k_range_resolution, 2.0 )*k_calibration*iq[offset_hh + i*n].x;
  float zdb = 10*log10( z );
  float zdr = 10*(log10( iq[offset_hh + i*n].x ) - log10( iq[offset_vv + i*n].x ));
  out[result_offset + i*2 + 0] = zdb;
  out[result_offset + i*2 + 1] = zdr;
}


void setup_ports() {
  subscriber.connect( "tcp://localhost:5563" );
  subscriber.setsockopt( ZMQ_SUBSCRIBE, "A", 1 );
  publisher.bind( "tcp://*:5563" );
}

void generate_hamming_coefficients(int m, int n) {
  cout << "Generating Hamming coefficients..." << endl;
  // Calculate normalization power on range cell
  float p_range = 0;
  for (int i = 0; i < m; i++) {
    p_range = p_range + pow( 0.53836 - 0.46164*cos( 2*M_PI*(i)/(m - 1)), 2.0 );
  }
  p_range = p_range/m;

  // Calculate normalization power on Doppler cell
  float p_doppler = 0;
  for (int j = 0; j < n; j++) {
    p_doppler = p_doppler + pow( 0.53836 - 0.46164*cos( 2*M_PI*(j)/(n - 1)), 2.0 );
  }
  p_doppler = p_doppler/n;

  // Constant since FFT is not normalized and the power is computed w.r.t. 50ohm
  const float k_wind = -1/(16383.5*m*n*sqrt( 50 ));
  const float c = k_wind/sqrt( p_range*p_doppler );

  // Generate elements
  hamming_coef = new float[m*n];
  for (int i = 0; i < m; i++) {
    for (int j = 0; j < n; j++) {
      hamming_coef[j + i*n] =
          (0.53836 - 0.46164*cos( 2*M_PI*(i)/(m - 1)))*(0.53836 - 0.46164*cos( 2*M_PI*(j)/(n - 1)))*c;
    }
  }
}

void generate_ma_coefficients(Dimension4 dim, int n) {
  cout << "Generating MA coefficients..." << endl;
  float *ma_coef = new float[n];
  float _sum = 0.0;
  for (int i = 0; i < n; i++) {
    ma_coef[i] = exp( -(pow( i - ((n - 1)/2), 2.0 ))/2 );
    _sum += ma_coef[i];
  }
  for (int i = 0; i < n; i++) {
    ma_coef[i] = ma_coef[i]/_sum;
  }

  fftwf_complex *_fft_ma = (fftwf_complex *) fftwf_malloc( sizeof( fftwf_complex )*dim.width );
  fftwf_plan fft_ma_plan = fftwf_plan_dft_1d( dim.width, _fft_ma, _fft_ma, FFTW_FORWARD, FFTW_ESTIMATE );
  for (int j = 0; j < n; j++) {
    _fft_ma[j][0] = ma_coef[j];
    _fft_ma[j][1] = 0;
  }
  for (int j = n; j < dim.width; j++) {
    _fft_ma[j][0] = 0;
    _fft_ma[j][1] = 0;
  }
  fftwf_execute( fft_ma_plan );
  fftwf_destroy_plan( fft_ma_plan );
  fft_ma = new hipFloatComplex[dim.width];
  for (int j = 0; j < dim.width; j++) {
    fft_ma[j] = make_hipFloatComplex( _fft_ma[j][0], _fft_ma[j][1] );
  }
  fftwf_free( _fft_ma );
}

void generate_constants(Dimension4 dim, int ma_count) {
  cout << "Generating constants..." << endl;
  generate_hamming_coefficients( dim.height, dim.width );
  generate_ma_coefficients( dim, ma_count );
}

void prepare_host_arys(Dimension4 idim, Dimension4 sitdim) {
  cout << "Preparing host arrays..." << endl;
  hipHostMalloc((void **) &p_iq, idim.total_size*sizeof( hipFloatComplex ));
  result = new float[sitdim.total_size];
}

void prepare_device_arys(Dimension4 idim, Dimension4 odim) {
  cout << "Preparing device arrays..." << endl;
  hipMalloc( &d_hamming, idim.m_size*sizeof( float ));
  hipMalloc( &d_iq, idim.total_size*sizeof( hipFloatComplex ));
  hipMalloc( &d_tmp, idim.m_size*sizeof( hipFloatComplex ));
  hipMalloc( &d_result, odim.total_size*sizeof( float ));

  hipMemcpy( d_hamming, hamming_coef, idim.m_size*sizeof( float ), hipMemcpyHostToDevice );
  hipMemcpyToSymbol(HIP_SYMBOL( d_ma), fft_ma, idim.width*sizeof( hipFloatComplex ), 0, hipMemcpyHostToDevice );
}

void prepare_arys(Dimension4 idim, Dimension4 odim, Dimension4 sitdim) {
  cout << "Preparing arrays:" << endl;
  prepare_host_arys( idim, sitdim );
  prepare_device_arys( idim, odim );
}

void initialize_streams(Dimension4 idim, Dimension4 odim) {
  cout << "Initializing streams..." << endl;
  fft_range_handle = new hipfftHandle[idim.depth];
  fft_doppler_handle = new hipfftHandle[idim.depth];
  fft_pdop_handle = new hipfftHandle[idim.depth];

  int rank = 1;                 // --- 1D FFTs
  int nn[] = { idim.height };   // --- Size of the Fourier transform
  int istride = idim.width,     // --- Distance between two successive input/output elements
  ostride = idim.width;
  int idist = 1, odist = 1;     // --- Distance between batches
  int inembed[] = { 0 };        // --- Input size with pitch (ignored for 1D transforms)
  int onembed[] = { 0 };        // --- Output size with pitch (ignored for 1D transforms)
  int batch = idim.width;       // --- Number of batched executions

  streams = new hipStream_t[idim.depth];
  for (int i = 0; i < idim.depth; i++) {
    hipStreamCreate( &streams[i] );

    hipfftPlanMany( &fft_range_handle[i], rank, nn,
                   inembed, istride, idist,
                   onembed, ostride, odist, HIPFFT_C2C, batch );
    hipfftPlan1d( &fft_doppler_handle[i], idim.width, HIPFFT_C2C, idim.height );
    hipfftPlan1d( &fft_pdop_handle[i], idim.width, HIPFFT_C2C, idim.height/2 );

    hipfftSetStream( fft_range_handle[i], streams[i] );
    hipfftSetStream( fft_doppler_handle[i], streams[i] );
    hipfftSetStream( fft_pdop_handle[i], streams[i] );
  }
}

void read_matrix(Dimension4 idim, int sector, int elevation, int stream) {
  cout << "Reading matrices from network..." << endl;
  
  //  Read envelope with address
  string address = s_recv( subscriber );
  //  Read message contents
  string str = s_recv( subscriber );

  msgpack::object_handle oh = msgpack::unpack( str.data(), str.size() );
  msgpack::object deserialized = oh.get();

  sectormsg s;
  deserialized.convert( s );

  int idx = 0;
#pragma unroll
  for (int j = 0; j < idim.height; j++) {
#pragma unroll
    for (int i = 0; i < idim.width; i++) {
      // cin >> a >> b;
      p_iq[idim.copy_at_depth( i, j, 0, stream )] = make_hipFloatComplex( s.i_hh[idx], s.q_hh[idx] );
      p_iq[idim.copy_at_depth( i, j, 1, stream )] = make_hipFloatComplex( s.i_vv[idx], s.q_vv[idx] );
      p_iq[idim.copy_at_depth( i, j, 2, stream )] = make_hipFloatComplex( s.i_vh[idx], s.q_vh[idx] );
      idx++;
    }
  }

//  for (int j = 0; j < idim.height; j++) {
//    for (int i = 0; i < idim.width; i++) {
//      int idx = idim.copy_at_depth( i, j, 0, stream );
//      cout << "(" << p_iq[idx].x << "," << p_iq[idx].y << ") ";
//    }
//    cout << endl;
//  }
//  exit( 0 );
}

void copy_matrix_to_device(Dimension4 idim, int sector, int elevation, int stream) {
  cout << "Copying matrices to device..." << endl;
  hipMemcpyAsync(
      &d_iq[idim.copy_at_depth( 0, 0, 0, stream )],
      &p_iq[idim.copy_at_depth( 0, 0, 0, stream )],
      idim.m_size*idim.copies*sizeof( hipFloatComplex ),
      hipMemcpyHostToDevice,
      streams[stream] );
}

void perform_stage_1(Dimension4 idim, int stream) {
  cout << "Performing Stage I..." << endl;

  int
      offset_hh = idim.copy_at_depth( 0, 0, 0, stream ),
      offset_vv = idim.copy_at_depth( 0, 0, 1, stream ),
      offset_vh = idim.copy_at_depth( 0, 0, 2, stream );

  // apply Hamming coefficients
  __apply_hamming<<<idim.height, idim.width, 0, streams[stream]>>>( d_iq, d_hamming, offset_hh );
  __apply_hamming<<<idim.height, idim.width, 0, streams[stream]>>>( d_iq, d_hamming, offset_vv );
  __apply_hamming<<<idim.height, idim.width, 0, streams[stream]>>>( d_iq, d_hamming, offset_vh );

  // FFT range profile
  hipfftExecC2C( fft_range_handle[stream], &d_iq[offset_hh], &d_iq[offset_hh], HIPFFT_FORWARD );
  hipfftExecC2C( fft_range_handle[stream], &d_iq[offset_vv], &d_iq[offset_vv], HIPFFT_FORWARD );
  hipfftExecC2C( fft_range_handle[stream], &d_iq[offset_vh], &d_iq[offset_vh], HIPFFT_FORWARD );

  // FFT+shift Doppler profile
  __sum_v4<<<idim.height/2, idim.width, 2*idim.width*sizeof(hipFloatComplex), streams[stream]>>>( d_iq, d_tmp,
                                                                                                 offset_hh );
  __avgconj<<<idim.height, idim.width, 0, streams[stream]>>>( d_iq, d_tmp, offset_hh );
  __sum_v4<<<idim.height/2, idim.width, 2*idim.width*sizeof(hipFloatComplex), streams[stream]>>>( d_iq, d_tmp,
                                                                                                 offset_vv );
  __avgconj<<<idim.height, idim.width, 0, streams[stream]>>>( d_iq, d_tmp, offset_vv );
  __sum_v4<<<idim.height/2, idim.width, 2*idim.width*sizeof(hipFloatComplex), streams[stream]>>>( d_iq, d_tmp,
                                                                                                 offset_vh );
  __avgconj<<<idim.height, idim.width, 0, streams[stream]>>>( d_iq, d_tmp, offset_vh );

  hipfftExecC2C( fft_doppler_handle[stream], &d_iq[offset_hh], &d_iq[offset_hh], HIPFFT_FORWARD );
  hipfftExecC2C( fft_doppler_handle[stream], &d_iq[offset_vv], &d_iq[offset_vv], HIPFFT_FORWARD );
  hipfftExecC2C( fft_doppler_handle[stream], &d_iq[offset_vh], &d_iq[offset_vh], HIPFFT_FORWARD );

  __conjugate<<<idim.height, idim.width, 0, streams[stream]>>>( d_iq, offset_hh );
  __conjugate<<<idim.height, idim.width, 0, streams[stream]>>>( d_iq, offset_vv );
  __conjugate<<<idim.height, idim.width, 0, streams[stream]>>>( d_iq, offset_vh );

  __shift<<<idim.height, idim.width/2, 0, streams[stream]>>>( d_iq, idim.width, offset_hh );
  __shift<<<idim.height, idim.width/2, 0, streams[stream]>>>( d_iq, idim.width, offset_vv );
  __shift<<<idim.height, idim.width/2, 0, streams[stream]>>>( d_iq, idim.width, offset_vh );

  __clip_v2<<<2, idim.height, 0, streams[stream]>>>( d_iq, idim.width, offset_hh );
  __clip_v2<<<2, idim.height, 0, streams[stream]>>>( d_iq, idim.width, offset_vv );
  __clip_v2<<<2, idim.height, 0, streams[stream]>>>( d_iq, idim.width, offset_vh );
}

void perform_stage_2(Dimension4 idim, int stream) {
  cout << "Performing Stage II..." << endl;

  int
      offset_hh = idim.copy_at_depth( 0, 0, 0, stream ),
      offset_vv = idim.copy_at_depth( 0, 0, 1, stream ),
      offset_vh = idim.copy_at_depth( 0, 0, 2, stream );

  // Get absolute value squared
  __abssqr<<<idim.height/2, idim.width, 0, streams[stream]>>>( d_iq, offset_hh );
  __abssqr<<<idim.height/2, idim.width, 0, streams[stream]>>>( d_iq, offset_vv );
  __abssqr<<<idim.height/2, idim.width, 0, streams[stream]>>>( d_iq, offset_vh );

  // FFT PDOP
  hipfftExecC2C( fft_pdop_handle[stream], &d_iq[offset_hh], &d_iq[offset_hh], HIPFFT_FORWARD );
  hipfftExecC2C( fft_pdop_handle[stream], &d_iq[offset_vv], &d_iq[offset_vv], HIPFFT_FORWARD );
  hipfftExecC2C( fft_pdop_handle[stream], &d_iq[offset_vh], &d_iq[offset_vh], HIPFFT_FORWARD );

  // Apply MA coefficients
  __apply_ma<<<idim.height/2, idim.width, 0, streams[stream]>>>( d_iq, offset_hh );
  __apply_ma<<<idim.height/2, idim.width, 0, streams[stream]>>>( d_iq, offset_vv );
  __apply_ma<<<idim.height/2, idim.width, 0, streams[stream]>>>( d_iq, offset_vh );

  // Inverse FFT
  hipfftExecC2C( fft_pdop_handle[stream], &d_iq[offset_hh], &d_iq[offset_hh], HIPFFT_BACKWARD );
  hipfftExecC2C( fft_pdop_handle[stream], &d_iq[offset_vv], &d_iq[offset_vv], HIPFFT_BACKWARD );
  hipfftExecC2C( fft_pdop_handle[stream], &d_iq[offset_vh], &d_iq[offset_vh], HIPFFT_BACKWARD );

  __scale_real<<<idim.height/2, idim.width, 0, streams[stream]>>>( d_iq, offset_hh );
  __scale_real<<<idim.height/2, idim.width, 0, streams[stream]>>>( d_iq, offset_vv );
  __scale_real<<<idim.height/2, idim.width, 0, streams[stream]>>>( d_iq, offset_vh );

  // Sum
  __sum_inplace_v4<<<idim.height/4, idim.width, 2*idim.width*sizeof(hipFloatComplex), streams[stream]>>>( d_iq,
                                                                                                         offset_hh );
  __sum_inplace_v4<<<idim.height/4, idim.width, 2*idim.width*sizeof(hipFloatComplex), streams[stream]>>>( d_iq,
                                                                                                         offset_vv );
  __sum_inplace_v4<<<idim.height/4, idim.width, 2*idim.width*sizeof(hipFloatComplex), streams[stream]>>>( d_iq,
                                                                                                         offset_vh );
}

void perform_stage_3(Dimension4 idim, Dimension4 odim, int sector, int elevation, int stream) {
  cout << "Performing Stage III..." << endl;

  int
      offset_hh = idim.copy_at_depth( 0, 0, 0, stream ),
      offset_vv = idim.copy_at_depth( 0, 0, 1, stream ),
      offset_vh = idim.copy_at_depth( 0, 0, 2, stream );

  // Calculate ZdB, Zdr
  __calcresult_v2<<<1, idim.height/2, 0, streams[stream]>>>(
      d_iq,
      d_result,
      idim.width,
      offset_hh, offset_vv, offset_vh,
      odim.copy_at_depth( 0, 0, 0, stream ));
}

void advance(Dimension4 idim) {
  cout << "Advancing to next sector..." << endl;
  current_sector = (current_sector + 1)%n_sectors;
  if (current_sector == 0) {
    current_elevation = (current_elevation + 1)%n_elevations;
  }
  current_stream = (current_stream + 1)%idim.depth;
}

void copy_result_to_host(Dimension4 idim, Dimension4 odim, Dimension4 sitdim, int sector, int elevation, int stream) {
/*
  cout << 1 << endl;
  hipFloatComplex *dump = new hipFloatComplex[idim.m_size];
  cout << 2 << endl;
  hipMemcpyAsync(
      dump,
      &d_iq[idim.copy_at_depth( 0, 0, 0, stream )],
      idim.m_size * sizeof( hipFloatComplex ),
      hipMemcpyDeviceToHost,
      streams[stream] );
  cout << 3 << endl;

  for (int j = 0; j < idim.height/2; j++) {
    for (int i = 0; i < idim.width; i++) {
      int idx = idim.copy_at_depth( i, j, 0, 0 );
      cout << "(" << dump[idx].x << "," << dump[idx].y << ") ";
    }
    cout << endl;
  }
  cout << 4 << endl;
  exit( 0 );
*/
  cout << "Copying result to host..." << endl;

  hipMemcpyAsync(
      &result[sitdim.copy_at_depth( 0, 0, sector, elevation )],
      &d_result[odim.copy_at_depth( 0, 0, 0, stream )],
      odim.m_size*sizeof( float ),
      hipMemcpyDeviceToHost,
      streams[stream] );

//  cout << "zdb:" << endl;
//  for (int i=0; i<sitdim.height; i++) {
//    cout << result[sitdim.copy_at_depth(0,i,sector,elevation)] << endl;
//  }
//  exit(0);
}

void send_results(Dimension4 sitdim, int sector, int elevation) {
  cout << "Sending results to network..." << endl;

  float *zdb = new float[sitdim.height];
  float *zdr = new float[sitdim.height];

  for (int i = 0; i < sitdim.height; i++) {
    zdb[i] = result[sitdim.copy_at_depth( 0, i, sector, elevation )];
    zdr[i] = result[sitdim.copy_at_depth( 1, i, sector, elevation )];
  }

  unsigned char *zdb_buff = new unsigned char[sizeof( float )*sitdim.height + 2]; // + 2 for sector id
  unsigned char *zdr_buff = new unsigned char[sizeof( float )*sitdim.height + 2]; // + 2 for sector id
  zdb_buff[0] = (sector >> 8) & 0xff;
  zdb_buff[1] = (sector) & 0xff;
  zdr_buff[0] = (sector >> 8) & 0xff;
  zdr_buff[1] = (sector) & 0xff;
  aftoab( zdb, sitdim.height, &zdb_buff[2] );
  aftoab( zdr, sitdim.height, &zdr_buff[2] );

  // zdb_client->send((const char *) zdb_buff, sitdim.height*sizeof( float ) + 2 );
  // zdr_client->send((const char *) zdr_buff, sitdim.height*sizeof( float ) + 2 );
}

void do_process(Dimension4 idim, Dimension4 odim, Dimension4 sitdim) {
  cout << "Starting main loop..." << endl;
  read_matrix( idim, current_sector, current_elevation, current_stream );
  copy_matrix_to_device( idim, current_sector, current_elevation, current_stream );
  do {
    perform_stage_1( idim, current_stream );
    perform_stage_2( idim, current_stream );
    perform_stage_3( idim, odim, current_sector, current_elevation, current_stream );
    int
        prev_sector = current_sector,
        prev_elevation = current_elevation,
        prev_stream = current_stream;
    advance( idim );
    read_matrix( idim, current_sector, current_elevation, current_stream );
    copy_matrix_to_device( idim, current_sector, current_elevation, current_stream );
    copy_result_to_host( idim, odim, sitdim, prev_sector, prev_elevation, prev_stream );
    send_results( sitdim, prev_sector, prev_elevation );
  } while (true);
}

void destroy_streams() {
  cout << "Destroying streams..." << endl;

}

void destroy_device_arys() {
  cout << "Destroying device arrays..." << endl;

}

void destroy_host_arys() {
  cout << "Destroying host arrays..." << endl;

}

void destroy_arrays() {
  cout << "Destroying arrays:" << endl;
  destroy_device_arys();
  destroy_host_arys();
}

int main(int argc, char **argv) {

  ios_base::sync_with_stdio( false );

  int num_streams = 2;
  if (argc > 1) {
    num_streams = atoi( argv[1] );
    num_streams = num_streams < 2 ? 2 : num_streams;
  }

  Dimension4 idim( n_samples, n_sweeps, 3, num_streams );
  Dimension4 odim( 2, n_sweeps/2, 1, num_streams );
  Dimension4 sitdim( 2, n_sweeps/2, n_sectors, n_elevations );

  setup_ports();

  generate_constants( idim, ma_count );
  prepare_arys( idim, odim, sitdim );
  initialize_streams( idim, odim );
  do_process( idim, odim, sitdim );
  destroy_streams();
  destroy_arrays();

  return 0;
}
