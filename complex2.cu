#include <iostream>
#include <stdlib.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>
#include <math.h>
#include <complex.h>
#include <hip/hip_complex.h>

using namespace std;

hipFloatComplex *pcmul(hipFloatComplex *a, float *b, int m, int n) {
    hipFloatComplex *res = new hipFloatComplex[m*n];

    float real, imag;
    for (int i=0; i<m; i++) {
        for (int j=0; j<n; j++) {
            real = hipCrealf(a[i*n+j]);
            imag = hipCimagf(a[i*n+j]);
            res[i*n+j] = make_hipFloatComplex(b[i*n+j]*real, b[i*n+j]*imag);
        }
    }

    return res;
}

__global__ void pcmul_kernel(hipFloatComplex *res, hipFloatComplex *a, float *b, int m, int n) {

    //unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i<m*n) {
        float real, imag;
        real = hipCrealf(a[i]);
        imag = hipCimagf(a[i]);
        res[i] = make_hipFloatComplex(b[i]*real, b[i]*imag);
    }
}

hipFloatComplex *pcmul_gpu(hipFloatComplex *a, float *b, int m, int n) {

    // host
    hipFloatComplex *h_res;
    // device
    hipFloatComplex *d_a, *d_res;
    float *d_b;

    struct timeval tb, te;
    unsigned long long bb, e;

    h_res = new hipFloatComplex[m*n];

    hipMalloc(&d_res, m*n*sizeof(hipFloatComplex));
    hipMalloc(&d_a, m*n*sizeof(hipFloatComplex));
    hipMalloc(&d_b, m*n*sizeof(float));

    gettimeofday(&tb, NULL);

    hipMemcpy(d_a, a, m*n*sizeof(hipFloatComplex), hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, m*n*sizeof(float), hipMemcpyHostToDevice);

    gettimeofday(&te, NULL);
    bb = (unsigned long long)(tb.tv_sec) * 1000000 + (unsigned long long)(tb.tv_usec) / 1;
    e = (unsigned long long)(te.tv_sec) * 1000000 + (unsigned long long)(te.tv_usec) / 1;

    cout << "copy to device " << e-bb << endl;

    gettimeofday(&tb, NULL);
    pcmul_kernel<<<m,n>>>(d_res, d_a, d_b, m, n);
    gettimeofday(&te, NULL);
    bb = (unsigned long long)(tb.tv_sec) * 1000000 + (unsigned long long)(tb.tv_usec) / 1;
    e = (unsigned long long)(te.tv_sec) * 1000000 + (unsigned long long)(te.tv_usec) / 1;

    cout << "kernel compute " << e-bb << endl;

    gettimeofday(&tb, NULL);
    hipMemcpy(h_res, d_res, m*n*sizeof(hipFloatComplex), hipMemcpyDeviceToHost);
    gettimeofday(&te, NULL);
    bb = (unsigned long long)(tb.tv_sec) * 1000000 + (unsigned long long)(tb.tv_usec) / 1;
    e = (unsigned long long)(te.tv_sec) * 1000000 + (unsigned long long)(te.tv_usec) / 1;

    cout << "copy to host " << e-bb << endl;

    hipFree(d_res);
    hipFree(d_a);
    hipFree(d_b);

    return h_res;
}

int main(int argc, char **argv) {

    int m, n;
    hipFloatComplex *iq, *mul;
    float *h;

    m = 1024;
    n = 512;

    iq = new hipFloatComplex[m*n];
    h = new float[m*n];

    for (int i=0; i<m; i++) {
        for (int j=0; j<n; j++) {
            iq[i*n+j] = make_hipFloatComplex((float)i, (float)j);
            h[i*n+j] = i;
            //cout << "(" << cuCrealf(iq[i*n+j]) << "," << cuCimagf(iq[i*n+j]) << ") ";
        }
        //cout << endl;
    }

    /*for (int i=0; i<m; i++) {
        for (int j=0; j<n; j++) {
            cout << h[i*n+j] << " ";
        }
        cout << endl;
    }*/

    struct timeval tb, te;
    unsigned long long b, e;

    gettimeofday(&tb, NULL);
    mul = pcmul(iq, h, m, n);
    gettimeofday(&te, NULL);
    b = (unsigned long long)(tb.tv_sec) * 1000000 + (unsigned long long)(tb.tv_usec) / 1;
    e = (unsigned long long)(te.tv_sec) * 1000000 + (unsigned long long)(te.tv_usec) / 1;

    cout << e-b << endl;

    gettimeofday(&tb, NULL);
    mul = pcmul_gpu(iq, h, m, n);
    gettimeofday(&te, NULL);
    b = (unsigned long long)(tb.tv_sec) * 1000000 + (unsigned long long)(tb.tv_usec) / 1;
    e = (unsigned long long)(te.tv_sec) * 1000000 + (unsigned long long)(te.tv_usec) / 1;

    cout << e-b << endl;

    /*for (int i=0; i<m; i++) {
        for (int j=0; j<n; j++) {
            cout << "(" << cuCrealf(mul[i*n+j]) << "," << cuCimagf(mul[i*n+j]) << ") ";
        }
        cout << endl;
    }*/

    delete[] iq;
    delete[] h;
    delete[] mul;

    return 0;
}
