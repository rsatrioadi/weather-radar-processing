#include "hip/hip_runtime.h"
#include <iostream>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hip/hip_complex.h>
#include <fftw3.h>
#include <hipfft/hipfft.h>
#include <sys/time.h>
#include <assert.h>

using namespace std;

#define k_rangeres 30
#define k_calib 1941.05

#define RESULT_SIZE 2

#define DEBUG

inline
hipError_t checkCuda(hipError_t result) {
#if defined(DEBUG) || defined(_DEBUG)
    if (result != hipSuccess) {
        fprintf(stderr, "CUDA Runtime Error: %s\n", hipGetErrorString(result));
        assert(result == hipSuccess);
    }
#endif
    return result;
}

float *generate_hamming_coef(int m, int n) {

    // Calculate normalization power on range cell
    float p_range=0;
    for(int i=0; i < m; i++) {
        p_range=p_range+pow(0.53836-0.46164*cos(2*M_PI*(i)/(m-1)), 2.0);
    }
    p_range=p_range/m;

    // Calculate normalization power on Doppler cell
    float p_doppler=0;
    for(int j=0; j < n; j++) {
        p_doppler=p_doppler+pow(0.53836-0.46164*cos(2*M_PI*(j)/(n-1)), 2.0);
    }
    p_doppler=p_doppler/n;

    // Constant since FFT is not normalized and the power is computed w.r.t. 50ohm
    const float K_wind = -1/(16383.5*m*n*sqrt(50));
    const float c = K_wind/sqrt(p_range*p_doppler);

    // Generate elements
    float *_hamming_coef= new float[m*n];
    for(int i=0; i < m; i++) {
        for(int j=0; j < n; j++) {
            _hamming_coef[i*n+j] = (0.53836-0.46164*cos(2*M_PI*(i)/(m-1))) * (0.53836-0.46164*cos(2*M_PI*(j)/(n-1))) * c;
        }
    }

    return _hamming_coef;
}

float *generate_ma_coef(int n){
    float *_ma_coef = new float[n];
    float _sum = 0.0;
    for(int i=0; i < n; i++) {
        _ma_coef[i]=exp(-(pow(i-((n-1)/2), 2.0))/2);
        _sum += _ma_coef[i];
    }
    for(int i=0; i < n; i++){
        _ma_coef[i] = _ma_coef[i]/_sum;
    }
    return _ma_coef;
}

__constant__ hipFloatComplex d_ma[512];

__global__ void __apply_hamming(hipFloatComplex *a, float *b) {
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    a[idx] = make_hipFloatComplex(b[idx]*hipCrealf(a[idx]), b[idx]*hipCimagf(a[idx]));
}

__global__ void __apply_ma(hipFloatComplex *inout, hipFloatComplex *macoef) {
    const unsigned int i = blockIdx.x, j = threadIdx.x, n = blockDim.x;

    inout[i*n+j] = hipCmulf(inout[i*n+j], macoef[j]);
}

__global__ void __apply_ma_v2(hipFloatComplex *inout) {
    const unsigned int i = blockIdx.x, j = threadIdx.x, n = blockDim.x;

    inout[i*n+j] = hipCmulf(inout[i*n+j], d_ma[j]);
}

__global__ void __conjugate(hipFloatComplex *a) {
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    a[idx].y *= -1;
}

__global__ void __shift(hipFloatComplex *inout, int n) {
    const unsigned int i = blockIdx.x, j = threadIdx.x;

    hipFloatComplex temp = inout[i*n+j];
    inout[i*n+j] = inout[i*n+(j+n/2)];
    inout[i*n+(j+n/2)] = temp;
}

__global__ void __clip(hipFloatComplex *inout, int n) {
    const unsigned int i = blockIdx.x, j = n-threadIdx.x-1;
    inout[i*n+j] = make_hipFloatComplex(0, 0);
}

__global__ void __abssqr(hipFloatComplex *inout, int n) {
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;

    float real, imag;
    real = hipCrealf(inout[idx]);
    imag = hipCimagf(inout[idx]);
    inout[idx] = make_hipFloatComplex(real*real + imag*imag, 0);    
}

__global__ void __sum(hipFloatComplex *in, hipFloatComplex *out) {
    const unsigned int i = blockIdx.x, j = threadIdx.x, n = blockDim.x;

    out[i*n+j] = make_hipFloatComplex(in[i*n+j].x, in[i*n+j].y);
    __syncthreads();

    for (unsigned int s=blockDim.x/2; s>0; s>>=1) {
        if (j < s) {
            out[i*n+j] = hipCaddf(out[i*n+j], out[i*n+j+s]);
        }
        __syncthreads();
    }
}

__global__ void __sum_v2(hipFloatComplex *in, hipFloatComplex *out) {
    const unsigned int i = 2*blockIdx.x, j = threadIdx.x, n = blockDim.x;

    #pragma unroll
    for (unsigned int d=0; d<2; d++) {
        out[i*n+j+n*d] = make_hipFloatComplex(in[i*n+j+n*d].x, in[i*n+j+n*d].y);
    }
    __syncthreads();

    for (unsigned int s=blockDim.x/2; s>0; s>>=1) {
        if (j < s) {
            #pragma unroll
            for (unsigned int d=0; d<2; d++) {
                out[i*n+j+n*d] = hipCaddf(out[i*n+j+n*d], out[i*n+j+n*d+s]);
            }
        }
        __syncthreads();
    }
}

__global__ void __sum_v3(hipFloatComplex *in, hipFloatComplex *out) {
    const unsigned int i = blockIdx.x, j = threadIdx.x, n = blockDim.x;
    extern __shared__ hipFloatComplex sdata[];

    sdata[j] = make_hipFloatComplex(in[i*n+j].x, in[i*n+j].y);
    __syncthreads();

    for (unsigned int s=blockDim.x/2; s>0; s>>=1) {
        if (j < s) {
            sdata[j] = hipCaddf(sdata[j], sdata[j+s]);
        }
        __syncthreads();
    }

    if(j==0) {
        out[i*n] = sdata[j];
    }
}

__global__ void __sum_v4(hipFloatComplex *in, hipFloatComplex *out) {
    const unsigned int i = 2*blockIdx.x, j = threadIdx.x, n = blockDim.x;
    extern __shared__ hipFloatComplex sdata[];

    #pragma unroll
    for (unsigned int d=0; d<2; d++) {
        sdata[j+n*d] = make_hipFloatComplex(in[i*n+j+n*d].x, in[i*n+j+n*d].y);
    }
    __syncthreads();

    for (unsigned int s=blockDim.x/2; s>0; s>>=1) {
        if (j < s) {
            #pragma unroll
            for (unsigned int d=0; d<2; d++) {
                sdata[j+n*d] = hipCaddf(sdata[j+n*d], sdata[j+n*d+s]);
            }
        }
        __syncthreads();
    }

    if(j==0) {
        #pragma unroll
        for (unsigned int d=0; d<2; d++) {
            out[i*n+n*d] = sdata[j+n*d];
        }
    }
}

__global__ void __sum_inplace(hipFloatComplex *g_idata) {
    const unsigned int i = blockIdx.x, j = threadIdx.x, n = blockDim.x;
    
    // __syncthreads();
    for (unsigned int s=blockDim.x/2; s>0; s>>=1) {
        if (j < s) {
            // g_idata[i] = make_hipFloatComplex(g_idata[i].x+g_idata[i + s].x, 0);
            g_idata[i*n+j] = hipCaddf(g_idata[i*n+j], g_idata[i*n+j+s]);
        }
        __syncthreads();
    }
}

__global__ void __sum_inplace_v2(hipFloatComplex *g_idata) {
    const unsigned int i = 2*blockIdx.x, j = threadIdx.x, n = blockDim.x;
    
    // __syncthreads();
    for (unsigned int s=blockDim.x/2; s>0; s>>=1) {
        if (j < s) {
            // g_idata[i] = make_hipFloatComplex(g_idata[i].x+g_idata[i + s].x, 0);
            #pragma unroll
            for (unsigned int d=0; d<2; d++) {
                g_idata[i*n+j+n*d] = hipCaddf(g_idata[i*n+j+n*d], g_idata[i*n+j+n*d+s]);
            }
        }
        __syncthreads();
    }
}

__global__ void __sum_inplace_v3(hipFloatComplex *in) {
    const unsigned int i = blockIdx.x, j = threadIdx.x, n = blockDim.x;
    extern __shared__ hipFloatComplex sdata[];

    sdata[j] = make_hipFloatComplex(in[i*n+j].x, in[i*n+j].y);
    __syncthreads();

    for (unsigned int s=blockDim.x/2; s>0; s>>=1) {
        if (j < s) {
            sdata[j] = hipCaddf(sdata[j], sdata[j+s]);
        }
        __syncthreads();
    }

    if(j==0) {
        in[i*n] = sdata[j];
    }
}

__global__ void __sum_inplace_v4(hipFloatComplex *in) {
    const unsigned int i = 2*blockIdx.x, j = threadIdx.x, n = blockDim.x;
    extern __shared__ hipFloatComplex sdata[];

    #pragma unroll
    for (unsigned int d=0; d<2; d++) {
        sdata[j+n*d] = make_hipFloatComplex(in[i*n+j+n*d].x, in[i*n+j+n*d].y);
    }
    __syncthreads();

    for (unsigned int s=blockDim.x/2; s>0; s>>=1) {
        if (j < s) {
            #pragma unroll
            for (unsigned int d=0; d<2; d++) {
                sdata[j+n*d] = hipCaddf(sdata[j+n*d], sdata[j+n*d+s]);
            }
        }
        __syncthreads();
    }

    if(j==0) {
        #pragma unroll
        for (unsigned int d=0; d<2; d++) {
            in[i*n+n*d] = sdata[j+n*d];
        }
    }
}

__global__ void __avgconj(hipFloatComplex *inout, hipFloatComplex *sum) {
    const unsigned int i = blockIdx.x, j = threadIdx.x, n = blockDim.x;

    float avgx = sum[i*n].x/n;
    float avgy = sum[i*n].y/n;
    inout[i*n+j] = make_hipFloatComplex(inout[i*n+j].x-avgx, (inout[i*n+j].y-avgy)*-1);
}

__global__ void __scale_real(hipFloatComplex *inout) {
    const unsigned int i = blockIdx.x, j = threadIdx.x, n = blockDim.x;

    inout[i*n+j] = make_hipFloatComplex(inout[i*n+j].x/n, 0);
}

__global__ void __calcresult(hipFloatComplex *hh, hipFloatComplex *vv, float *out, int n) {
    const unsigned int i = blockIdx.x;

    float z = pow(i*k_rangeres, 2.0) * k_calib * hh[i*n].x;
    float zdb = 10 * log10(z);
    float zdr = 10 * (log10(hh[i*n].x)-log10(vv[i*n].x));
    out[i*RESULT_SIZE+0] = zdb;
    out[i*RESULT_SIZE+1] = zdr;
}

__global__ void __calcresult_v2(hipFloatComplex *hh, hipFloatComplex *vv, float *out, int n) {
    const unsigned int i = threadIdx.x;

    float z = pow(i*k_rangeres, 2.0) * k_calib * hh[i*n].x;
    float zdb = 10 * log10(z);
    float zdr = 10 * (log10(hh[i*n].x)-log10(vv[i*n].x));
    out[i*RESULT_SIZE+0] = zdb;
    out[i*RESULT_SIZE+1] = zdr;
}

void tick(timeval *begin) {
    gettimeofday(begin, NULL);
}

void tock(timeval *begin, timeval *end, string caption) {
    unsigned long long bb, e;

    gettimeofday(end, NULL);
    bb = (unsigned long long)(begin->tv_sec) * 1000000 + (unsigned long long)(begin->tv_usec) / 1;
    e = (unsigned long long)(end->tv_sec) * 1000000 + (unsigned long long)(end->tv_usec) / 1;

    cout << caption << ": " << e-bb << endl;
}

int main(int argc, char **argv) {
    ios_base::sync_with_stdio(false);

    struct timeval tb, te;

    tick(&tb);

    hipFloatComplex *iqhh, *iqvv;
    float *result;
    int sector_id;

    const int m = 1024; // cell
    const int n = 512;  // sweep

    const int ma_count = 7;

    iqhh = new hipFloatComplex[m*n];
    iqvv = new hipFloatComplex[m*n];
    result = new float[(m/2)*RESULT_SIZE];

    float a, b;

    // Generate Hamming coefficients
    const float *hamming_coef = generate_hamming_coef(m, n);

    // Generate MA coefficients
    float *ma_coef = generate_ma_coef(ma_count);
    fftwf_complex *_fft_ma = (fftwf_complex*) fftwf_malloc(sizeof(fftwf_complex) * n);
    fftwf_plan fft_ma_plan = fftwf_plan_dft_1d(n, _fft_ma, _fft_ma, FFTW_FORWARD, FFTW_ESTIMATE);
    for (int j=0; j<ma_count; j++) {
        _fft_ma[j][0] = ma_coef[j];
        _fft_ma[j][1] = 0;
    }
    for (int j=ma_count; j<n; j++) {
        _fft_ma[j][0] = 0;
        _fft_ma[j][1] = 0;
    }
    fftwf_execute(fft_ma_plan);
    fftwf_destroy_plan(fft_ma_plan);
    hipFloatComplex *fft_ma;
    fft_ma = new hipFloatComplex[n];
    for (int j=0; j<n; j++) {
        fft_ma[j] = make_hipFloatComplex(_fft_ma[j][0], _fft_ma[j][1]);
    }
    fftwf_free(_fft_ma);

    // Device buffers
    /*__constant__*/ float *d_hamming;
    // /*__constant__*/ hipFloatComplex *d_ma;
    hipFloatComplex *d_iqhh, *d_iqvv;
    hipFloatComplex *d_sum;
    float *d_result;
    //float *d_powhh, *d_powvv;

    hipMalloc(&d_hamming, m*n*sizeof(float));
    // hipMalloc(&d_ma, n*sizeof(hipFloatComplex));
    hipMalloc(&d_iqhh, m*n*sizeof(hipFloatComplex));
    hipMalloc(&d_iqvv, m*n*sizeof(hipFloatComplex));
    hipMalloc(&d_sum, m*n*sizeof(hipFloatComplex));
    hipMalloc(&d_result, (m/2)*RESULT_SIZE*sizeof(float));

    hipMemcpy(d_hamming, hamming_coef, m*n*sizeof(float), hipMemcpyHostToDevice);
    // hipMemcpy(d_ma, fft_ma, n*sizeof(hipFloatComplex), hipMemcpyHostToDevice);
    hipMemcpyToSymbol(HIP_SYMBOL(d_ma), fft_ma, n*sizeof(hipFloatComplex), 0, hipMemcpyHostToDevice);

    // CUFFT initialization
    hipfftHandle fft_range_handle;
    hipfftHandle fft_doppler_handle;
    hipfftHandle fft_pdop_handle;

    int rank = 1;                   // --- 1D FFTs
    int nn[] = { m };               // --- Size of the Fourier transform
    int istride = n, ostride = n;   // --- Distance between two successive input/output elements
    int idist = 1, odist = 1;       // --- Distance between batches
    int inembed[] = { 0 };          // --- Input size with pitch (ignored for 1D transforms)
    int onembed[] = { 0 };          // --- Output size with pitch (ignored for 1D transforms)
    int batch = n;                  // --- Number of batched executions

    hipfftPlanMany(&fft_range_handle, rank, nn, 
                  inembed, istride, idist,
                  onembed, ostride, odist, HIPFFT_C2C, batch);
    hipfftPlan1d(&fft_doppler_handle, n, HIPFFT_C2C, m);
    hipfftPlan1d(&fft_pdop_handle, n, HIPFFT_C2C, m/2);

    tock(&tb, &te, "initialization");

    float ms; // elapsed time in milliseconds

    sector_id = -1;

    // create events and streams
    hipEvent_t startEvent, stopEvent;

    hipEventCreate(&startEvent);
    hipEventCreate(&stopEvent);
    // hipEventCreate(&dummyEvent);

    hipEventRecord(startEvent,0);
    tick(&tb);

    while(sector_id < 126) {

        // tick(&tb);

        // Read 1 sector data
        // cin >> sector_id;
        // sector_id++;
        for (int i=0; i<m; i++) {
            for (int j=0; j<n; j++) {
                // cin >> a >> b;
                iqhh[i*n+j] = make_hipFloatComplex(i, j);
            }
        }
        for (int i=0; i<m; i++) {
            for (int j=0; j<n; j++) {
                // cin >> a >> b;
                iqvv[i*n+j] = make_hipFloatComplex(j, i);
            }
        }

        hipMemcpy(d_iqhh, iqhh, m*n*sizeof(hipFloatComplex), hipMemcpyHostToDevice);
        hipMemcpy(d_iqvv, iqvv, m*n*sizeof(hipFloatComplex), hipMemcpyHostToDevice);

        // apply Hamming coefficients
        __apply_hamming<<<m,n>>>(d_iqhh, d_hamming);
        __apply_hamming<<<m,n>>>(d_iqvv, d_hamming);

        // FFT range profile
        hipfftExecC2C(fft_range_handle, d_iqhh, d_iqhh, HIPFFT_FORWARD);
        hipfftExecC2C(fft_range_handle, d_iqvv, d_iqvv, HIPFFT_FORWARD);

        // FFT+shift Doppler profile
        __sum_v4<<<m/2,n,2*n*sizeof(hipFloatComplex)>>>(d_iqhh, d_sum);
        __avgconj<<<m,n>>>(d_iqhh, d_sum);
        __sum_v4<<<m/2,n,2*n*sizeof(hipFloatComplex)>>>(d_iqvv, d_sum);
        __avgconj<<<m,n>>>(d_iqvv, d_sum);

        hipfftExecC2C(fft_doppler_handle, d_iqhh, d_iqhh, HIPFFT_FORWARD);
        hipfftExecC2C(fft_doppler_handle, d_iqvv, d_iqvv, HIPFFT_FORWARD);

        __conjugate<<<m,n>>>(d_iqhh);
        __conjugate<<<m,n>>>(d_iqvv);

        __shift<<<m,n/2>>>(d_iqhh, n);
        __shift<<<m,n/2>>>(d_iqvv, n);

        __clip<<<m,2>>>(d_iqhh, n);
        __clip<<<m,2>>>(d_iqvv, n);

        // Get absolute value
        __abssqr<<<m/2,n>>>(d_iqhh, n);
        __abssqr<<<m/2,n>>>(d_iqvv, n);

        // FFT PDOP
        hipfftExecC2C(fft_pdop_handle, d_iqhh, d_iqhh, HIPFFT_FORWARD);
        hipfftExecC2C(fft_pdop_handle, d_iqvv, d_iqvv, HIPFFT_FORWARD);

        // Apply MA coefficients
        __apply_ma_v2<<<m/2,n>>>(d_iqhh);
        __apply_ma_v2<<<m/2,n>>>(d_iqvv);

        // Inverse FFT
        hipfftExecC2C(fft_pdop_handle, d_iqhh, d_iqhh, HIPFFT_BACKWARD);
        hipfftExecC2C(fft_pdop_handle, d_iqvv, d_iqvv, HIPFFT_BACKWARD);

        __scale_real<<<m/2,n>>>(d_iqhh);
        __scale_real<<<m/2,n>>>(d_iqvv);

        // Sum
        __sum_inplace_v4<<<m/4,n,2*n*sizeof(hipFloatComplex)>>>(d_iqhh);
        __sum_inplace_v4<<<m/4,n,2*n*sizeof(hipFloatComplex)>>>(d_iqvv);

        // hipMemcpy(iqhh, d_iqhh, m*n*sizeof(hipFloatComplex), hipMemcpyDeviceToHost);
        // hipMemcpy(iqvv, d_iqvv, m*n*sizeof(hipFloatComplex), hipMemcpyDeviceToHost);

        // for (int i=0; i<m/2; i++) {
        //     float z = pow(i*k_rangeres, 2.0) * k_calib * iqhh[i*n].x;
        //     float zdb = 10 * log10(z);
        //     float zdr = 10 * (log10(iqhh[i*n].x)-log10(iqvv[i*n].x));
        //     cout << zdb << " " << zdr << endl;
        // }
        // exit(0);

        // Calculate ZdB, Zdr
        __calcresult_v2<<<1,m/2>>>(d_iqhh, d_iqvv, d_result, n);

        hipMemcpy(result, d_result, (m/2)*RESULT_SIZE*sizeof(float), hipMemcpyDeviceToHost);

        // for (int i=0; i<m/2; i++) {
        //     for (int j=0; j<RESULT_SIZE; j++) {
        //         cout << result[i*RESULT_SIZE+j] << " ";
        //     }
        //     cout << endl;
        // }
        // exit(0);
    }

    tock(&tb, &te, "All (us)");

    hipEventRecord(stopEvent, 0);
    hipEventSynchronize(stopEvent);
    hipEventElapsedTime(&ms, startEvent, stopEvent);
    printf("Time for sequential transfer and execute (ms): %f\n", ms);

    hipEventDestroy(startEvent);
    hipEventDestroy(stopEvent);

    hipFree(d_hamming);
    hipFree(d_ma);
    hipFree(d_iqhh);
    hipFree(d_iqvv);

    delete[] iqhh;
    delete[] iqvv;

    return 0;
}

    // hipMemcpy(iqhh, d_iqhh, m*n*sizeof(hipFloatComplex), hipMemcpyDeviceToHost);
    // hipMemcpy(iqvv, d_iqvv, m*n*sizeof(hipFloatComplex), hipMemcpyDeviceToHost);

    // for (int i=0; i<m; i++) {
    //     for (int j=0; j<n; j++) {
    //         cout << "(" << iqhh[i*n+j].x << "," << iqhh[i*n+j].y << ") ";
    //     }
    //     cout << endl;
    // }
    // // for (int i=0; i<m; i++) {
    // //     for (int j=0; j<n; j++) {
    // //         cout << iqvv[i*n+j].x << " ";
    // //     }
    // //     cout << endl;
    // // }
    // exit(0);
