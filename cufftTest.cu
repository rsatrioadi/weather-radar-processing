
#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>

#include <iostream>
#include <complex>

// #define DATA_LEN 1024
// #define ITERATION 100000

int main(int argc, char **argv)
{
	if (argc != 3)
	{
		std::cout << "Usage: " << argv[0] << " [DATA_LEN] [ITERATION]" << std::endl;
		return 1;
	}
	int DATA_LEN = atoi(argv[1]);
	int ITERATION = atoi(argv[2]);
	// Siapkan memory untuk input data di Host
	// cufftComplex *t_HostInputData = (cufftComplex*)malloc(sizeof(cufftComplex)*DATA_LEN*1);
	hipfftComplex *t_HostInputData;
	hipError_t status = hipHostMalloc((void**)&t_HostInputData, sizeof(hipfftComplex)*DATA_LEN*1, hipHostMallocDefault);
	for(int i=0; i < DATA_LEN; i++)
	{
		t_HostInputData[i].x = 1.0;
		t_HostInputData[i].y = 1.0;
	}

	// for(int i=0; i < DATA_LEN; i++)
	// {
	// 	std::cout << t_HostInputData[i].x << " + i" << t_HostInputData[i].y << std::endl;
	// }
	// std::cout << std::endl;

	// Siapkan memory untuk data di GPU
	hipfftComplex *t_InputData;
	hipfftComplex *t_OutputData;
	hipMalloc((void**)&t_InputData, sizeof(hipfftComplex)*DATA_LEN*1);
	hipMalloc((void**)&t_OutputData, sizeof(hipfftComplex)*DATA_LEN*1);
	if (hipGetLastError() != hipSuccess)
	{
		std::cout << "Cuda error: Failed to allocate" << std::endl;
		return 1;
	}
	hipMemset(t_InputData, 0, DATA_LEN);
	hipMemcpy(t_InputData, t_HostInputData, sizeof(hipfftComplex)*DATA_LEN*1, hipMemcpyHostToDevice);
	hipMemset(t_OutputData, 0, DATA_LEN);

	// FFT plan
	hipfftHandle t_Plan;
	if (hipfftPlan1d(&t_Plan, DATA_LEN, HIPFFT_C2C, 1) != HIPFFT_SUCCESS)
	{
			std::cout << "CUFFT error: Plan creation failed" << std::endl;
			return 1;
	}

	hipEvent_t start, end;
	hipEventCreate(&start);
	hipEventCreate(&end);
	float elapsedTime;
	hipEventRecord(start, 0);

	// Execute FFT Forward operation
	for(int i=0; i < ITERATION; i++)
	{
		if (hipfftExecC2C(t_Plan, t_InputData, t_OutputData, HIPFFT_FORWARD) != HIPFFT_SUCCESS)
		{
			std::cout << "CUFFT error: ExecC2C Forward failed" << std::endl;
			return 1;
		}
	}

	hipEventRecord(end, 0);
	hipEventSynchronize(end);
	hipEventElapsedTime(&elapsedTime, start, end);
	printf("%d times for the FFT: %fms\n", ITERATION, elapsedTime);

	// // Execute FFT Backward / IFFT operation
	// if (cufftExecC2C(t_Plan, t_OutputData, t_InputData, CUFFT_INVERSE) != CUFFT_SUCCESS)
	// {
	// 	std::cout << "CUFFT error: ExecC2C Forward failed" << std::endl;
	// 	return 1;
	// }

	// Synchro
	if (hipDeviceSynchronize() != hipSuccess)
	{
		std::cout << "Cuda error: Failed to synchronize" << std::endl;
		return 1;
	}

	// Siapkan host memory untuk menerima result FFT dari GPU
	hipfftComplex *t_HostData = (hipfftComplex*)malloc(sizeof(hipfftComplex)*DATA_LEN*1);

	// Copy from GPU to host memroy
	hipMemcpy(t_HostData, t_OutputData, sizeof(hipfftComplex)*DATA_LEN*1, hipMemcpyDeviceToHost);

	// Display data
	// for(int i=0; i < DATA_LEN; i++)
	// {
	// 	std::cout << t_HostData[i].x << " + i" << t_HostData[i].y << std::endl;
	// }

	// Cleaning stuff
	hipfftDestroy(t_Plan);
	hipFree(t_InputData);
	hipFree(t_OutputData);
	hipHostFree(t_HostInputData);

	return 0;
}
