#include <iostream>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hip/hip_complex.h>

using namespace std;

__global__ void __sumcomplex(hipDoubleComplex *g_idata, hipDoubleComplex *g_odata) {
    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    g_odata[i] = make_hipDoubleComplex(g_idata[i].x, g_idata[i].y);
    __syncthreads();
    for (unsigned int s=blockDim.x/2; s>0; s>>=1) {
        if (tid < s) {
            g_odata[i] = make_hipDoubleComplex(g_odata[i].x+g_odata[i + s].x, g_odata[i].y+g_odata[i + s].y);
        }
        __syncthreads();
    }
}

__global__ void __sum_inplace(hipDoubleComplex *g_idata) {
    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    __syncthreads();
    for (unsigned int s=blockDim.x/2; s>0; s>>=1) {
        if (tid < s) {
            g_idata[i] = make_hipDoubleComplex(g_idata[i].x+g_idata[i + s].x, 0);
        }
        __syncthreads();
    }
}

int main() {

	const int m=16, n=8;

	hipDoubleComplex *h_data = new hipDoubleComplex[m*n];
	hipDoubleComplex *h_sum = new hipDoubleComplex[m*n];
	hipDoubleComplex *d_data, *d_sum;

    hipMalloc(&d_data, m*n*sizeof(hipDoubleComplex));
    hipMalloc(&d_sum, m*n*sizeof(hipDoubleComplex));

    for (int i=0; i<m; i++) {
	    for (int j=0; j<n; j++) {
	    	h_data[i*n+j] = make_hipDoubleComplex(i+j, 0);
	    }
    }

    cout << "in:" << endl;
    for (int i=0; i<m; i++) {
	    for (int j=0; j<n; j++) {
			cout << "(" << h_data[i*n+j].x << "," << h_data[i*n+j].y << ") ";
		}
	    cout << endl;
    }

    hipMemcpy(d_data, h_data, m*n*sizeof(hipDoubleComplex), hipMemcpyHostToDevice);
    __sum_inplace<<<m,n>>>(d_data);
    hipMemcpy(h_data, d_data, m*n*sizeof(hipDoubleComplex), hipMemcpyDeviceToHost);

    cout << "out:" << endl;
    for (int i=0; i<m; i++) {
	    for (int j=0; j<n; j++) {
	    	cout << "(" << h_data[i*n+j].x << "," << h_data[i*n+j].y << ") ";
	    }
	    cout << endl;   
    }

	return 0;
}
