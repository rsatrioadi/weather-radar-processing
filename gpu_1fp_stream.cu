#include "hip/hip_runtime.h"
#include <iostream>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hip/hip_complex.h>
#include <fftw3.h>
#include <hipfft/hipfft.h>
#include <sys/time.h>
#include <assert.h>

using namespace std;

#define k_rangeres 30
#define k_calib 1941.05

#define RESULT_SIZE 2

//#define NSTREAMS 16

#define DEBUG

inline
hipError_t checkCuda(hipError_t result) {
#if defined(DEBUG) || defined(_DEBUG)
    if (result != hipSuccess) {
        fprintf(stderr, "CUDA Runtime Error: %s\n", hipGetErrorString(result));
        assert(result == hipSuccess);
    }
#endif
    return result;
}

float *generate_hamming_coef(int m, int n) {

    // Calculate normalization power on range cell
    float p_range=0;
    for(int i=0; i < m; i++) {
        p_range=p_range+pow(0.53836-0.46164*cos(2*M_PI*(i)/(m-1)), 2.0);
    }
    p_range=p_range/m;

    // Calculate normalization power on Doppler cell
    float p_doppler=0;
    for(int j=0; j < n; j++) {
        p_doppler=p_doppler+pow(0.53836-0.46164*cos(2*M_PI*(j)/(n-1)), 2.0);
    }
    p_doppler=p_doppler/n;

    // Constant since FFT is not normalized and the power is computed w.r.t. 50ohm
    const float K_wind = -1/(16383.5*m*n*sqrt(50));
    const float c = K_wind/sqrt(p_range*p_doppler);

    // Generate elements
    float *_hamming_coef= new float[m*n];
    for(int i=0; i < m; i++) {
        for(int j=0; j < n; j++) {
            _hamming_coef[i*n+j] = (0.53836-0.46164*cos(2*M_PI*(i)/(m-1))) * (0.53836-0.46164*cos(2*M_PI*(j)/(n-1))) * c;
        }
    }

    return _hamming_coef;
}

float *generate_ma_coef(int n){
    float *_ma_coef = new float[n];
    float _sum = 0.0;
    for(int i=0; i < n; i++) {
        _ma_coef[i]=exp(-(pow(i-((n-1)/2), 2.0))/2);
        _sum += _ma_coef[i];
    }
    for(int i=0; i < n; i++){
        _ma_coef[i] = _ma_coef[i]/_sum;
    }
    return _ma_coef;
}

__global__ void __apply_hamming(hipFloatComplex *a, float *b, int offset) {
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    a[offset+idx] = make_hipFloatComplex(b[idx]*hipCrealf(a[offset+idx]), b[idx]*hipCimagf(a[offset+idx]));
}

__global__ void __sum(hipFloatComplex *g_idata, hipFloatComplex *g_odata, int offset) {
    unsigned int i = blockIdx.x, j = threadIdx.x, n = blockDim.x;

    g_odata[offset+i*n+j] = make_hipFloatComplex(g_idata[offset+i*n+j].x, g_idata[offset+i*n+j].y);
    __syncthreads();

    for (unsigned int s=blockDim.x/2; s>0; s>>=1) {
        if (j < s) {
            g_odata[offset+i*n+j] = hipCaddf(g_odata[offset+i*n+j], g_odata[offset+i*n+j+s]);
        }
        __syncthreads();
    }
}

__global__ void __avgconj(hipFloatComplex *g_idata, hipFloatComplex *g_odata, int offset) {
    unsigned int i = blockIdx.x, j = threadIdx.x, n = blockDim.x;

    float avgx = g_odata[offset+i*n].x/n;
    float avgy = g_odata[offset+i*n].y/n;
    g_idata[offset+i*n+j] = make_hipFloatComplex(g_idata[offset+i*n+j].x-avgx, (g_idata[offset+i*n+j].y-avgy)*-1);
}

__global__ void __conjugate(hipFloatComplex *a, int offset) {
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    a[offset+idx].y *= -1;
}

__global__ void __shift(hipFloatComplex *inout, int n, int offset) {
    unsigned int i = blockIdx.x, j = threadIdx.x;

    hipFloatComplex temp = inout[offset+i*n+j];
    inout[offset+i*n+j] = inout[offset+i*n+(j+n/2)];
    inout[offset+i*n+(j+n/2)] = temp;
}

__global__ void __clip(hipFloatComplex *inout, int n, int offset) {
    unsigned int i = blockIdx.x, j = n-threadIdx.x-1;
    inout[offset+i*n+j] = make_hipFloatComplex(0, 0);
}

__global__ void __abssqr(hipFloatComplex *inout, int n, int offset) {
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;

    float real, imag;
    real = hipCrealf(inout[offset+idx]);
    imag = hipCimagf(inout[offset+idx]);
    inout[offset+idx] = make_hipFloatComplex(real*real + imag*imag, 0);    
}

__global__ void __apply_ma(hipFloatComplex *inout, hipFloatComplex *macoef, int offset) {
    unsigned int i = blockIdx.x, j = threadIdx.x, n = blockDim.x;

    inout[offset+i*n+j] = hipCmulf(inout[offset+i*n+j], macoef[j]);
}

__global__ void __scale_real(hipFloatComplex *inout, int offset) {
    unsigned int i = blockIdx.x, j = threadIdx.x, n = blockDim.x;

    inout[offset+i*n+j] = make_hipFloatComplex(inout[offset+i*n+j].x/n, 0);
}

__global__ void __sum_inplace(hipFloatComplex *g_idata, int offset) {
    unsigned int i = blockIdx.x, j = threadIdx.x, n = blockDim.x;
    
    __syncthreads();
    for (unsigned int s=blockDim.x/2; s>0; s>>=1) {
        if (j < s) {
            // g_idata[i] = make_hipFloatComplex(g_idata[i].x+g_idata[i + s].x, 0);
            g_idata[offset+i*n+j] = hipCaddf(g_idata[offset+i*n+j], g_idata[offset+i*n+j+s]);
        }
        __syncthreads();
    }
}

__global__ void __calcresult(hipFloatComplex *hh, hipFloatComplex *vv, float *out, int n, int offset, int result_offset) {
    unsigned int i = blockIdx.x;

    float z = pow(i*k_rangeres, 2.0) * k_calib * hh[offset+i*n].x;
    float zdb = 10 * log10(z);
    float zdr = 10 * (log10(hh[offset+i*n].x)-log10(vv[offset+i*n].x));
    out[result_offset+i*RESULT_SIZE+0] = zdb;
    out[result_offset+i*RESULT_SIZE+1] = zdr;
}

void tick(timeval *begin) {
    gettimeofday(begin, NULL);
}

void tock(timeval *begin, timeval *end, string caption) {
    unsigned long long bb, e;

    gettimeofday(end, NULL);
    bb = (unsigned long long)(begin->tv_sec) * 1000000 + (unsigned long long)(begin->tv_usec) / 1;
    e = (unsigned long long)(end->tv_sec) * 1000000 + (unsigned long long)(end->tv_usec) / 1;

    cout << caption << ": " << e-bb << endl;
}

int main(int argc, char **argv) {
    ios_base::sync_with_stdio(false);

    int NSTREAMS = atoi(argv[1]);
    if (NSTREAMS == 0) {
        NSTREAMS = 1;
    }

    struct timeval tb, te;

    tick(&tb);

    hipFloatComplex *iqhh, *iqvv;
    float *result;
    int sector_id;

    const int m = 1024; // cell
    const int n = 512;  // sweep

    const int ma_count = 7;

    iqhh = new hipFloatComplex[NSTREAMS*m*n];
    iqvv = new hipFloatComplex[NSTREAMS*m*n];
    result = new float[NSTREAMS*(m/2)*RESULT_SIZE];

    float a, b;

    // Generate Hamming coefficients
    const float *hamming_coef = generate_hamming_coef(m, n);

    // Generate MA coefficients
    float *ma_coef = generate_ma_coef(ma_count);
    fftwf_complex *_fft_ma = (fftwf_complex*) fftwf_malloc(sizeof(fftwf_complex) * n);
    fftwf_plan fft_ma_plan = fftwf_plan_dft_1d(n, _fft_ma, _fft_ma, FFTW_FORWARD, FFTW_ESTIMATE);
    for (int j=0; j<ma_count; j++) {
        _fft_ma[j][0] = ma_coef[j];
        _fft_ma[j][1] = 0;
    }
    for (int j=ma_count; j<n; j++) {
        _fft_ma[j][0] = 0;
        _fft_ma[j][1] = 0;
    }
    fftwf_execute(fft_ma_plan);
    fftwf_destroy_plan(fft_ma_plan);
    hipFloatComplex *fft_ma;
    fft_ma = new hipFloatComplex[n];
    for (int j=0; j<n; j++) {
        fft_ma[j] = make_hipFloatComplex(_fft_ma[j][0], _fft_ma[j][1]);
    }
    fftwf_free(_fft_ma);

    // Device buffers
    /*__constant__*/ float *d_hamming;
    /*__constant__*/ hipFloatComplex *d_ma;
    hipFloatComplex *d_iqhh, *d_iqvv;
    hipFloatComplex *d_sum;
    float *d_result;
    //float *d_powhh, *d_powvv;

    hipMalloc(&d_hamming, m*n*sizeof(float));
    hipMalloc(&d_ma, n*sizeof(hipFloatComplex));
    hipMalloc(&d_iqhh, NSTREAMS*m*n*sizeof(hipFloatComplex));
    hipMalloc(&d_iqvv, NSTREAMS*m*n*sizeof(hipFloatComplex));
    hipMalloc(&d_sum, NSTREAMS*m*n*sizeof(hipFloatComplex));
    hipMalloc(&d_result, NSTREAMS*(m/2)*RESULT_SIZE*sizeof(float));

    hipMemcpy(d_hamming, hamming_coef, m*n*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_ma, fft_ma, n*sizeof(hipFloatComplex), hipMemcpyHostToDevice);

    // CUFFT initialization
    hipfftHandle *fft_range_handle = new hipfftHandle[NSTREAMS];
    hipfftHandle *fft_doppler_handle = new hipfftHandle[NSTREAMS];
    hipfftHandle *fft_pdop_handle = new hipfftHandle[NSTREAMS];

    int rank = 1;                   // --- 1D FFTs
    int nn[] = { m };               // --- Size of the Fourier transform
    int istride = n, ostride = n;   // --- Distance between two successive input/output elements
    int idist = 1, odist = 1;       // --- Distance between batches
    int inembed[] = { 0 };          // --- Input size with pitch (ignored for 1D transforms)
    int onembed[] = { 0 };          // --- Output size with pitch (ignored for 1D transforms)
    int batch = n;                  // --- Number of batched executions

    tock(&tb, &te, "initialization");

    float ms; // elapsed time in milliseconds

    sector_id = 0;

    // create events and streams
    hipEvent_t startEvent, stopEvent;
    hipStream_t stream[NSTREAMS];

    hipEventCreate(&startEvent);
    hipEventCreate(&stopEvent);
    // hipEventCreate(&dummyEvent);

    for (int i = 0; i < NSTREAMS; i++) {
        hipStreamCreate(&stream[i]);

        hipfftPlanMany(&fft_range_handle[i], rank, nn, 
                      inembed, istride, idist,
                      onembed, ostride, odist, HIPFFT_C2C, batch);
        hipfftPlan1d(&fft_doppler_handle[i], n, HIPFFT_C2C, m);
        hipfftPlan1d(&fft_pdop_handle[i], n, HIPFFT_C2C, m/2);

        hipfftSetStream(fft_range_handle[i], stream[i]);
        hipfftSetStream(fft_doppler_handle[i], stream[i]);
        hipfftSetStream(fft_pdop_handle[i], stream[i]);
    }

    hipEventRecord(startEvent,0);

    while(sector_id < 126) {

        // tick(&tb);

        // Read 1 sector data
        cin >> sector_id;

        int stream_id = sector_id % NSTREAMS;
        int offset = stream_id * (m*n);
        int result_offset = stream_id * (m/2)*RESULT_SIZE;

        for (int i=0; i<m; i++) {
            for (int j=0; j<n; j++) {
                cin >> a >> b;
                iqhh[offset+i*n+j] = make_hipFloatComplex(a, b);
            }
        }
        for (int i=0; i<m; i++) {
            for (int j=0; j<n; j++) {
                cin >> a >> b;
                iqvv[offset+i*n+j] = make_hipFloatComplex(a, b);
            }
        }

        // tock(&tb, &te, "read");

        // cout << "Processing sector " << sector_id << ", stream " << stream_id << endl;


        // for (int i=0; i<m; i++) {
        //     for (int j=0; j<n; j++) {
        //         cout << "(" << iqhh[i*n+j].x << "," << iqhh[i*n+j].y << ") ";
        //     }
        //     cout << endl;
        // }
        // // for (int i=0; i<m; i++) {
        // //     for (int j=0; j<n; j++) {
        // //         cout << iqvv[i*n+j].x << " ";
        // //     }
        // //     cout << endl;
        // // }
        // exit(0);
        
        // tick(&tb);

        hipMemcpyAsync(&d_iqhh[offset], &iqhh[offset], m*n*sizeof(hipFloatComplex), hipMemcpyHostToDevice, stream[stream_id]);
        hipMemcpyAsync(&d_iqvv[offset], &iqvv[offset], m*n*sizeof(hipFloatComplex), hipMemcpyHostToDevice, stream[stream_id]);

        // apply Hamming coefficients
        __apply_hamming<<<m,n,0,stream[stream_id]>>>(d_iqhh, d_hamming, offset);
        __apply_hamming<<<m,n,0,stream[stream_id]>>>(d_iqvv, d_hamming, offset);

        // FFT range profile
        hipfftExecC2C(fft_range_handle[stream_id], &d_iqhh[offset], &d_iqhh[offset], HIPFFT_FORWARD);
        hipfftExecC2C(fft_range_handle[stream_id], &d_iqvv[offset], &d_iqvv[offset], HIPFFT_FORWARD);

        // FFT+shift Doppler profile
        __sum<<<m,n,0,stream[stream_id]>>>(d_iqhh, d_sum, offset);
        __avgconj<<<m,n,0,stream[stream_id]>>>(d_iqhh, d_sum, offset);
        __sum<<<m,n,0,stream[stream_id]>>>(d_iqvv, d_sum, offset);
        __avgconj<<<m,n,0,stream[stream_id]>>>(d_iqvv, d_sum, offset);

        hipfftExecC2C(fft_doppler_handle[stream_id], &d_iqhh[offset], &d_iqhh[offset], HIPFFT_FORWARD);
        hipfftExecC2C(fft_doppler_handle[stream_id], &d_iqvv[offset], &d_iqvv[offset], HIPFFT_FORWARD);

        __conjugate<<<m,n,0,stream[stream_id]>>>(d_iqhh, offset);
        __conjugate<<<m,n,0,stream[stream_id]>>>(d_iqvv, offset);

        __shift<<<m,n/2,0,stream[stream_id]>>>(d_iqhh, n, offset);
        __shift<<<m,n/2,0,stream[stream_id]>>>(d_iqvv, n, offset);

        __clip<<<m,2,0,stream[stream_id]>>>(d_iqhh, n, offset);
        __clip<<<m,2,0,stream[stream_id]>>>(d_iqvv, n, offset);

        // Get absolute value squared
        __abssqr<<<m/2,n,0,stream[stream_id]>>>(d_iqhh, n, offset);
        __abssqr<<<m/2,n,0,stream[stream_id]>>>(d_iqvv, n, offset);

        // FFT PDOP
        hipfftExecC2C(fft_pdop_handle[stream_id], &d_iqhh[offset], &d_iqhh[offset], HIPFFT_FORWARD);
        hipfftExecC2C(fft_pdop_handle[stream_id], &d_iqvv[offset], &d_iqvv[offset], HIPFFT_FORWARD);

        // Apply MA coefficients
        __apply_ma<<<m/2,n,0,stream[stream_id]>>>(d_iqhh, d_ma, offset);
        __apply_ma<<<m/2,n,0,stream[stream_id]>>>(d_iqvv, d_ma, offset);

        // Inverse FFT
        hipfftExecC2C(fft_pdop_handle[stream_id], &d_iqhh[offset], &d_iqhh[offset], HIPFFT_BACKWARD);
        hipfftExecC2C(fft_pdop_handle[stream_id], &d_iqvv[offset], &d_iqvv[offset], HIPFFT_BACKWARD);

        __scale_real<<<m/2,n,0,stream[stream_id]>>>(d_iqhh, offset);
        __scale_real<<<m/2,n,0,stream[stream_id]>>>(d_iqvv, offset);

        // Sum
        __sum_inplace<<<m/2,n,0,stream[stream_id]>>>(d_iqhh, offset);
        __sum_inplace<<<m/2,n,0,stream[stream_id]>>>(d_iqvv, offset);

        // hipMemcpy(iqhh, d_iqhh, m*n*sizeof(hipFloatComplex), hipMemcpyDeviceToHost);
        // hipMemcpy(iqvv, d_iqvv, m*n*sizeof(hipFloatComplex), hipMemcpyDeviceToHost);

        // for (int i=0; i<m/2; i++) {
        //     float z = pow(i*k_rangeres, 2.0) * k_calib * iqhh[i*n].x;
        //     float zdb = 10 * log10(z);
        //     float zdr = 10 * (log10(iqhh[i*n].x)-log10(iqvv[i*n].x));
        //     cout << zdb << " " << zdr << endl;
        // }
        // exit(0);

        // Calculate ZdB, Zdr
        __calcresult<<<m/2,1,0,stream[stream_id]>>>(d_iqhh, d_iqvv, d_result, n, offset, result_offset);

        // hipDeviceSynchronize();
        // tock(&tb, &te, "processing");

        // tick(&tb);

        hipMemcpyAsync(&result[result_offset], &d_result[result_offset], (m/2)*RESULT_SIZE*sizeof(float), hipMemcpyDeviceToHost, stream[stream_id]);

        // tock(&tb, &te, "time");

        // for (int i=0; i<m/2; i++) {
        //     for (int j=0; j<RESULT_SIZE; j++) {
        //         cout << result[result_offset+i*RESULT_SIZE+j] << " ";
        //     }
        //     cout << endl;
        // }
    }

    hipEventRecord(stopEvent, 0);
    hipEventSynchronize(stopEvent);
    hipEventElapsedTime(&ms, startEvent, stopEvent);
    printf("Time for async transfer and execute (ms): %f\n", ms);

    hipEventDestroy(startEvent);
    hipEventDestroy(stopEvent);

    delete[] fft_range_handle;
    delete[] fft_doppler_handle;
    delete[] fft_pdop_handle;

    for (int i = 0; i < NSTREAMS; i++) {
        hipStreamDestroy(stream[i]);
    }

    hipFree(d_hamming);
    hipFree(d_ma);
    hipFree(d_iqhh);
    hipFree(d_iqvv);

    delete[] iqhh;
    delete[] iqvv;

    return 0;
}

    // hipMemcpy(iqhh, d_iqhh, m*n*sizeof(hipFloatComplex), hipMemcpyDeviceToHost);
    // hipMemcpy(iqvv, d_iqvv, m*n*sizeof(hipFloatComplex), hipMemcpyDeviceToHost);

    // for (int i=0; i<m; i++) {
    //     for (int j=0; j<n; j++) {
    //         cout << "(" << iqhh[i*n+j].x << "," << iqhh[i*n+j].y << ") ";
    //     }
    //     cout << endl;
    // }
    // // for (int i=0; i<m; i++) {
    // //     for (int j=0; j<n; j++) {
    // //         cout << iqvv[i*n+j].x << " ";
    // //     }
    // //     cout << endl;
    // // }
    // exit(0);
