#include "hip/hip_runtime.h"
#include <iostream>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hip/hip_complex.h>
#include <fftw3.h>
#include <hipfft/hipfft.h>

using namespace std;

double *generate_hamming_coef(int m, int n) {

    // Calculate normalization power on range cell
    double p_range=0;
    for(int i=0; i < m; i++) {
        p_range=p_range+pow(0.53836-0.46164*cos(2*M_PI*(i)/(m-1)), 2.0);
    }
    p_range=p_range/m;

    // Calculate normalization power on Doppler cell
    double p_doppler=0;
    for(int j=0; j < n; j++) {
        p_doppler=p_doppler+pow(0.53836-0.46164*cos(2*M_PI*(j)/(n-1)), 2.0);
    }
    p_doppler=p_doppler/n;

    // Constant since FFT is not normalized and the power is computed w.r.t. 50ohm
    const double K_wind = -1/(16383.5*m*n*sqrt(50));
    const double c = K_wind/sqrt(p_range*p_doppler);

    // Generate elements
    double *_hamming_coef= new double[m*n];
    for(int i=0; i < m; i++) {
        for(int j=0; j < n; j++) {
            _hamming_coef[i*n+j] = (0.53836-0.46164*cos(2*M_PI*(i)/(m-1))) * (0.53836-0.46164*cos(2*M_PI*(j)/(n-1))) * c;
        }
    }

    return _hamming_coef;
}

double *generate_ma_coef(int n){
    double *_ma_coef = new double[n];
    double _sum = 0.0;
    for(int i=0; i < n; i++) {
        _ma_coef[i]=exp(-(pow(i-((n-1)/2), 2.0))/2);
        _sum += _ma_coef[i];
    }
    for(int i=0; i < n; i++){
        _ma_coef[i] = _ma_coef[i]/_sum;
    }
    return _ma_coef;
}

__global__ void __apply_hamming(hipDoubleComplex *a, double *b, int m, int n) {
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;

    double real, imag;
    real = a[i].x;
    imag = a[i].y;
    a[i] = make_hipDoubleComplex(b[i%(m*n)]*real, b[i%(m*n)]*imag);
}

__global__ void __shift(hipDoubleComplex *out, hipDoubleComplex *in, int n) {
    unsigned int i = blockIdx.x, j = threadIdx.x;

    out[i*n+j] = make_hipDoubleComplex(in[i*n+(n/2-j)].x, in[i*n+(n/2-j)].y);
    out[i*n+j+n/2] = make_hipDoubleComplex(in[i*n+(n-j)].x, in[i*n+(n-j)].y);
}

int main(int argc, char **argv) {

    hipDoubleComplex *iq;
    double *pow_;

    const int m = 1024; // cell
    const int n = 512;  // sweep

    const int ma_count = 7;

    const double k_rangeres = 30;
    const double k_calib = 1941.05;

    iq = new hipDoubleComplex[2*m*n];
    pow_ = new double[m*n];

    double a, b;

    // Generate Hamming coefficients
    const double *hamming_coef = generate_hamming_coef(m, n);

    // Generate MA coefficients
    double *ma_coef = generate_ma_coef(ma_count);
    fftw_complex *_fft_ma = (fftw_complex*) fftw_malloc(sizeof(fftw_complex) * n);
    fftw_plan fft_ma_plan = fftw_plan_dft_1d(n, _fft_ma, _fft_ma, FFTW_FORWARD, FFTW_ESTIMATE);
    for (int j=0; j<ma_count; j++) {
        _fft_ma[j][0] = ma_coef[j];
        _fft_ma[j][1] = 0;
    }
    for (int j=ma_count; j<n; j++) {
        _fft_ma[j][0] = 0;
        _fft_ma[j][1] = 0;
    }
    fftw_execute(fft_ma_plan);
    fftw_destroy_plan(fft_ma_plan);

    hipDoubleComplex *fft_ma;
    fft_ma = new hipDoubleComplex[n];

    for (int j=0; j<n; j++) {
        fft_ma[j] = make_hipDoubleComplex(_fft_ma[j][0], _fft_ma[j][1]);
    }

    fftw_free(_fft_ma);

    // Device buffers
    /*__constant__*/ double *d_hamming;
    /*__constant__*/ hipDoubleComplex *d_ma;
    hipDoubleComplex *d_iq;
    hipDoubleComplex *d_shift;
    //double *d_pow;

    hipMalloc(&d_hamming, m*n*sizeof(double));
    hipMemcpy(d_hamming, hamming_coef, m*n*sizeof(double), hipMemcpyHostToDevice);
    hipMalloc(&d_ma, n*sizeof(hipDoubleComplex));
    hipMemcpy(d_ma, fft_ma, n*sizeof(hipDoubleComplex), hipMemcpyHostToDevice);
    hipMalloc(&d_iq, 2*m*n*sizeof(hipDoubleComplex));
    hipMalloc(&d_shift, 2*m*n*sizeof(hipDoubleComplex));

    // Read 1 sector data
    for (int i=0; i<m*2; i++) {
        for (int j=0; j<n; j++) {
            cin >> a >> b;
            iq[i*n+j] = make_hipDoubleComplex(a, b);
        }
    }

    // apply Hamming coefficients
    hipMemcpy(d_iq, iq, 2*m*n*sizeof(hipDoubleComplex), hipMemcpyHostToDevice);
    __apply_hamming<<<2*m,n>>>(d_iq, d_hamming, m, n);
    hipMemcpy(iq, d_iq, 2*m*n*sizeof(hipDoubleComplex), hipMemcpyDeviceToHost);

    // FFT range profile
    fftw_complex *fft_range_buffer;
    fftw_plan fft_range_plan;
    fft_range_buffer = (fftw_complex*) fftw_malloc(sizeof(fftw_complex) * m);
    fft_range_plan = fftw_plan_dft_1d(m, fft_range_buffer, fft_range_buffer, FFTW_FORWARD, FFTW_ESTIMATE);
    for (int j=0; j<n; j++) {

        // HH
        for (int i=0; i<m; i++) {
            fft_range_buffer[i][0] = iq[i*n+j].x;
            fft_range_buffer[i][1] = iq[i*n+j].y;
        }
        fftw_execute(fft_range_plan);
        for (int i=0; i<m; i++) {
            iq[i*n+j] = make_hipDoubleComplex(fft_range_buffer[i][0], fft_range_buffer[i][1]);
        }

        // VV
        for (int i=0; i<m; i++) {
            fft_range_buffer[i][0] = iq[(i+m)*n+j].x;
            fft_range_buffer[i][1] = iq[(i+m)*n+j].y;
        }
        fftw_execute(fft_range_plan);
        for (int i=0; i<m; i++) {
            iq[(i+m)*n+j] = make_hipDoubleComplex(fft_range_buffer[i][0], fft_range_buffer[i][1]);
        }
    }
    fftw_destroy_plan(fft_range_plan);
    fftw_free(fft_range_buffer);

    // FFT Doppler profile
    hipMemcpy(d_iq, iq, 2*m*n*sizeof(hipDoubleComplex), hipMemcpyHostToDevice);
    hipfftHandle handle;
    int rank = 1;                     // --- 1D FFTs
    int nn[] = { n };                 // --- Size of the Fourier transform
    int istride = 1, ostride = 1;     // --- Distance between two successive input/output elements
    int idist = n, odist = n;         // --- Distance between batches
    int inembed[] = { 0 };            // --- Input size with pitch (ignored for 1D transforms)
    int onembed[] = { 0 };            // --- Output size with pitch (ignored for 1D transforms)
    int batch = 2*m;                  // --- Number of batched executions
    hipfftPlanMany(&handle, rank, nn, 
                  inembed, istride, idist,
                  onembed, ostride, odist, HIPFFT_Z2Z, batch);
    //hipfftPlan1d(&handle, n, HIPFFT_Z2Z, 2*m);
    hipfftExecZ2Z(handle,  d_iq, d_iq, HIPFFT_FORWARD);
    __shift<<<2*m,n/2>>>(d_shift, d_iq, n);
    hipMemcpy(iq, d_shift, 2*m*n*sizeof(hipDoubleComplex), hipMemcpyDeviceToHost);
    for (int i=0; i<2*m; i++) {
        iq[i*n+(n-1)] = make_hipDoubleComplex(0,0);
        iq[i*n+(n-2)] = make_hipDoubleComplex(0,0);
    }

    // for (int i=0; i<m*2; i++) {
    //     for (int j=0; j<n; j++) {
    //         cout << "(" << iq[i*n+j].x << "," << iq[i*n+j].y << ") ";
    //     }
    //     cout << endl;
    // }
    // exit(0);

    hipDoubleComplex *iqhalf;
    iqhalf = new hipDoubleComplex[m*n];
    for (int i=0; i<m/2; i++) {
        for (int j=0; j<n; j++) {
            iqhalf[i*n+j] = make_hipDoubleComplex(iq[i*n+j].x, iq[i*n+j].y);
            iqhalf[(i+m/2)*n+j] = make_hipDoubleComplex(iq[(i+m)*n+j].x, iq[(i+m)*n+j].y);
        }
    }

    // PDOP
    fftw_complex *fft_pdop_buffer;
    fft_pdop_buffer = (fftw_complex*) fftw_malloc(sizeof(fftw_complex) * n);
    fftw_complex *fft_mult_buffer;
    fft_mult_buffer = (fftw_complex*) fftw_malloc(sizeof(fftw_complex) * n);
    fftw_plan fft_pdop_plan;
    fft_pdop_plan = fftw_plan_dft_1d(n, fft_pdop_buffer, fft_pdop_buffer, FFTW_FORWARD, FFTW_ESTIMATE);
    fftw_plan ifft_conv_plan;
    ifft_conv_plan = fftw_plan_dft_1d(n, fft_mult_buffer, fft_mult_buffer, FFTW_BACKWARD, FFTW_ESTIMATE);
    for (int i=0; i<m; i++) {

        for (int j=0; j<n; j++) {
            fft_pdop_buffer[j][0] = iqhalf[i*n+j].x * iqhalf[i*n+j].x + iqhalf[i*n+j].y * iqhalf[i*n+j].y;
            fft_pdop_buffer[j][1] = 0;
            //cout << fft_pdop_buffer[j][0] << " ";
        }
        //cout << endl;
        fftw_execute(fft_pdop_plan);
        for (int j=0; j<n; j++) {
            fft_mult_buffer[j][0] = fft_pdop_buffer[j][0] * fft_ma[j].x - fft_pdop_buffer[j][1] * fft_ma[j].y;
            fft_mult_buffer[j][1] = fft_pdop_buffer[j][0] * fft_ma[j].y + fft_pdop_buffer[j][1] * fft_ma[j].x;
            //cout << "(" << fft_mult_buffer[j][0] << "," << fft_mult_buffer[j][1] << ") ";
        }
        //cout << endl;
        fftw_execute(ifft_conv_plan);
        for (int j=0; j<n; j++) {
            pow_[i*n+j] = fft_mult_buffer[j][0]/n;
            //cout << pow_[i*n+j] << " ";
        }
        //cout << endl;
    }
    fftw_destroy_plan(ifft_conv_plan);
    fftw_destroy_plan(fft_pdop_plan);

    // Reflectivity
    double *z, *zdb, *zdr;
    z = new double[m];
    zdb = new double[m];
    zdr = new double[m/2];
    for (int i=0; i<m; i++) {
        for (int j=1; j<n; j++) {
            pow_[i*n] += pow_[i*n+j];
        }
        //cout << pow_[i*n] << endl;
    }
    for (int i=0; i<m/2; i++) {
        z[i] = pow(i*k_rangeres, 2.0) * k_calib * pow_[i*n];
        z[i+m/2] = pow(i*k_rangeres, 2.0) * k_calib * pow_[(i+m/2)*n];
        zdb[i] = 10 * log10(z[i]);
        //zdb[i+m/2] = 10 * log10(z[i+m/2]);
        zdr[i] = 10 * (log10(pow_[i*n])-log10(pow_[(i+m/2)*n]));
        cout << zdb[i] << " " << zdr[i] << endl;
    }

    hipFree(d_hamming);
    //hipFree(d_ma);
    hipFree(d_iq);
    hipFree(d_shift);

    delete iqhalf;

    delete zdr;
    delete zdb;
    delete z;

    fftw_free(fft_mult_buffer);
    fftw_free(fft_pdop_buffer);

    delete pow_;
    delete iq;

    return 0;
}
